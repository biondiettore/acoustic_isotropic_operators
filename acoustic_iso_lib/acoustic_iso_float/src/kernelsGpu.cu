#include "hip/hip_runtime.h"
#include "varDeclare.h"
#include <stdio.h>

/****************************************************************************************/
/***************************************** Debug shit ***********************************/
/****************************************************************************************/
__global__ void copyToWavefield(float *wavefield, float *slice, int its){

	int izGlobal = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // Global z-coordinate
	int ixGlobal = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // Global x-coordinate
	int iGlobal = dev_nz * ixGlobal + izGlobal; // 1D array index for the model on the global memory
	int iGlobalWavefield = its * dev_nz * dev_nx + iGlobal;
	wavefield[iGlobalWavefield] += slice[iGlobal];
}

__global__ void sumData(float *data, float *data1, float *data2, int nData){

	int iTime = blockIdx.x * BLOCK_SIZE + threadIdx.x; // Time coordinates
	int iRec = blockIdx.y * BLOCK_SIZE + threadIdx.y; // Receiver coordinates
	int iData = dev_nts * iRec + iTime; // 1D array index for the model on the global memory
	if (iData < nData){
		data[iData] = data1[iData] + data2[iData];
	}
}

__global__ void sumModels(float *model, float *model1, float *model2){

	int izGlobal = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // Global z-coordinate
	int ixGlobal = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // Global x-coordinate
	int iGlobal = dev_nz * ixGlobal + izGlobal; // 1D array index for the model on the global memory

	model[iGlobal] = model1[iGlobal] + model2[iGlobal];

}

/* Extract and interpolate data */
__global__ void recordInterpDataFine(float *dev_newTimeSlice, float *dev_signalOut, int itw, int *dev_receiversPositionReg) {
	int iThread = blockIdx.x * blockDim.x + threadIdx.x;
	if (iThread < dev_nReceiversReg) {
		dev_signalOut[dev_ntw*iThread+itw] += dev_newTimeSlice[dev_receiversPositionReg[iThread]];
	}
}

__global__ void interpWavefieldDebug(float *dev_wavefield, float *dev_timeSlice, int its, int it2, float *dev_vel2Dtw2In) {

	int izGlobal = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x;
	int ixGlobal = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y;
	int iGlobal = dev_nz * ixGlobal + izGlobal;
	int iGlobalWavefield = its * dev_nz * dev_nx + iGlobal;
	dev_wavefield[iGlobalWavefield] += dev_timeSlice[iGlobal] * dev_interpFilter[it2] * dev_vel2Dtw2In[iGlobal];
	dev_wavefield[iGlobalWavefield+dev_nz*dev_nx] += dev_timeSlice[iGlobal] * dev_interpFilter[dev_hInterpFilter+it2] * dev_vel2Dtw2In[iGlobal];

}

__global__ void scaleReflectivityDebug(float *dev_model, float *dev_reflectivityScaleIn){

	int izGlobal = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // Global z-coordinate
	int ixGlobal = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // Global x-coordinate
	int iGlobal = dev_nz * ixGlobal + izGlobal; // 1D array index for the model on the global memory
	dev_model[iGlobal] *= dev_reflectivityScaleIn[iGlobal];
}

__global__ void copyValueScaleDebug(float *dev_wavefield, float *dev_timeSlice, float *dev_vel2Dtw2In, int its) {

	long long izGlobal = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // Global z-coordinate
	long long ixGlobal = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // Global x-coordinate
	long long iGlobal = dev_nz * ixGlobal + izGlobal; // 1D array index for the model on the global memory
	long long iGlobalWavefield = its * dev_nz * dev_nx + iGlobal;
	dev_wavefield[iGlobalWavefield] = dev_timeSlice[iGlobal];// * dev_vel2Dtw2In[iGlobal];

}

__global__ void interpWavefieldScale(float *dev_wavefield, float *dev_timeSlice, float *dev_vel2Dtw2, int its, int it2) {

	int izGlobal = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // Global z-coordinate
	int ixGlobal = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // Global x-coordinate
	int iGlobal = dev_nz * ixGlobal + izGlobal; // 1D array index for the model on the global memory
	int iGlobalWavefield = its * dev_nz * dev_nx + iGlobal;
	dev_wavefield[iGlobalWavefield] += dev_timeSlice[iGlobal] * dev_interpFilter[it2] * dev_vel2Dtw2[iGlobal]; // its
	dev_wavefield[iGlobalWavefield+dev_nz*dev_nx] += dev_timeSlice[iGlobal] * dev_interpFilter[dev_hInterpFilter+it2] * dev_vel2Dtw2[iGlobal]; // its+1

}

/****************************************************************************************/
/***************************************** Injection ************************************/
/****************************************************************************************/
/* Inject source: no need for a "if" statement because the number of threads = nb devices */
__global__ void injectSource(float *dev_signalIn, float *dev_timeSlice, int itw, int *dev_sourcesPositionReg){
	int iThread = blockIdx.x * blockDim.x + threadIdx.x;
	dev_timeSlice[dev_sourcesPositionReg[iThread]] += dev_signalIn[iThread * dev_ntw + itw]; // Time is the fast axis
}

/* Interpolate and inject data */
__global__ void interpInjectData(float *dev_signalIn, float *dev_timeSlice, int its, int it2, int *dev_receiversPositionReg) {
	int iThread = blockIdx.x * blockDim.x + threadIdx.x;
	if (iThread < dev_nReceiversReg) {
		dev_timeSlice[dev_receiversPositionReg[iThread]] += dev_signalIn[dev_nts*iThread+its] * dev_interpFilter[it2+1] + dev_signalIn[dev_nts*iThread+its+1] * dev_interpFilter[dev_hInterpFilter+it2+1];
	}
}

/* Interpolate and inject secondary source at fine time-sampling */
__global__ void injectSecondarySource(float *dev_ssLeft, float *dev_ssRight, float *dev_p0, int indexFilter){
	int izGlobal = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // Global z-coordinate
	int ixGlobal = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // Global x-coordinate
	int iGlobal = dev_nz * ixGlobal + izGlobal; // 1D array index for the model on the global memory
	dev_p0[iGlobal] += dev_ssLeft[iGlobal] * dev_interpFilter[indexFilter] + dev_ssRight[iGlobal] * dev_interpFilter[dev_hInterpFilter+indexFilter];
}

/****************************************************************************************/
/*************************************** Extraction *************************************/
/****************************************************************************************/
/* Extract source for "nonlinear adjoint" */
__global__ void recordSource(float *dev_wavefield, float *dev_signalOut, int itw, int *dev_sourcesPositionReg) {
	int iThread = blockIdx.x * blockDim.x + threadIdx.x;
	dev_signalOut[dev_ntw*iThread + itw] += dev_wavefield[dev_sourcesPositionReg[iThread]];
}

/* Extract and interpolate data */
__global__ void recordInterpData(float *dev_newTimeSlice, float *dev_signalOut, int its, int it2, int *dev_receiversPositionReg) {

	int iThread = blockIdx.x * blockDim.x + threadIdx.x;
	if (iThread < dev_nReceiversReg) {
		// printf("dev_receiversPositionReg[iThread] = %d \n", dev_receiversPositionReg[iThread]);
		dev_signalOut[dev_nts*iThread+its]   += dev_newTimeSlice[dev_receiversPositionReg[iThread]] * dev_interpFilter[it2];
		dev_signalOut[dev_nts*iThread+its+1] += dev_newTimeSlice[dev_receiversPositionReg[iThread]] * dev_interpFilter[dev_hInterpFilter+it2];
	}
}

/****************************************************************************************/
/******************************** Wavefield extractions *********************************/
/****************************************************************************************/
__global__ void interpWavefield(float *dev_wavefield, float *dev_timeSlice, int its, int it2) {

	int izGlobal = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // Global z-coordinate
	int ixGlobal = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // Global x-coordinate
	int iGlobal = dev_nz * ixGlobal + izGlobal; // 1D array index for the model on the global memory
	int iGlobalWavefield = its * dev_nz * dev_nx + iGlobal;
	dev_wavefield[iGlobalWavefield] += dev_timeSlice[iGlobal] * dev_interpFilter[it2]; // its
	dev_wavefield[iGlobalWavefield+dev_nz*dev_nx] += dev_timeSlice[iGlobal] * dev_interpFilter[dev_hInterpFilter+it2]; // its+1

}

// Extract and scale receiver wavefield.
// Both scalings are done simultaneously: it works for time-lags extension, but not for subsurface offsets extension
__global__ void recordScaleWavefield(float *dev_wavefield, float *dev_timeSlice, int its, float *dev_reflectivityScale, float *dev_vel2Dtw2) {

	int izGlobal = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // Global z-coordinate
	int ixGlobal = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // Global x-coordinate
	int iGlobal = dev_nz * ixGlobal + izGlobal; // 1D array index for the model on the global memory
	long long iGlobalWavefield = its * dev_nz * dev_nx + iGlobal;

	dev_wavefield[iGlobalWavefield] += dev_timeSlice[iGlobal] * dev_reflectivityScale[iGlobal] * dev_vel2Dtw2[iGlobal];
}

// Simply records the receiver wavefield without applying any scaling (this is used for the subsurface offsets extension)
__global__ void recordWavefield(float *dev_wavefield, float *dev_timeSlice, int its) {

	int izGlobal = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // Global z-coordinate
	int ixGlobal = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // Global x-coordinate
	int iGlobal = dev_nz * ixGlobal + izGlobal; // 1D array index for the model on the global memory
	long long iGlobalWavefield = its * dev_nz * dev_nx + iGlobal;

	dev_wavefield[iGlobalWavefield] += dev_timeSlice[iGlobal];
}

__global__ void extractInterpAdjointWavefield(float *dev_timeSliceLeft, float *dev_timeSliceRight, float *dev_timeSliceFine, int it2) {

	int izGlobal = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // Global z-coordinate
	int ixGlobal = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // Global x-coordinate
	int iGlobal = dev_nz * ixGlobal + izGlobal; // 1D array index for the model on the global memory
	dev_timeSliceLeft[iGlobal]  += dev_timeSliceFine[iGlobal] * dev_interpFilter[it2]; // its
	dev_timeSliceRight[iGlobal] += dev_timeSliceFine[iGlobal] * dev_interpFilter[dev_hInterpFilter+it2]; // its+1
}

__global__ void interpFineToCoarseSlice(float *dev_timeSliceLeft, float *dev_timeSliceRight, float *dev_timeSliceFine, int it2) {

	int izGlobal = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // Global z-coordinate
	int ixGlobal = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // Global x-coordinate
	int iGlobal = dev_nz * ixGlobal + izGlobal; // 1D array index for the model on the global memory
	dev_timeSliceLeft[iGlobal]  += dev_timeSliceFine[iGlobal] * dev_interpFilter[it2]; // its
	dev_timeSliceRight[iGlobal] += dev_timeSliceFine[iGlobal] * dev_interpFilter[dev_hInterpFilter+it2]; // its+1
}

/****************************************************************************************/
/************************************ Time derivative ***********************************/
/****************************************************************************************/

__global__ void srcWfldSecondTimeDerivative(float *dev_wavefield, float *dev_slice0, float *dev_slice1, float *dev_slice2, int its) {

	int izGlobal = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // Global z-coordinate
	int ixGlobal = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // Global x-coordinate
	int iGlobal = dev_nz * ixGlobal + izGlobal; // 1D array index for the model on the global memory
	int iGlobalWavefield = its * dev_nz * dev_nx + iGlobal;

	// Apply second time derivative
	dev_wavefield[iGlobalWavefield] = dev_cSide * ( dev_slice0[iGlobal] + dev_slice2[iGlobal] ) + dev_cCenter * dev_slice1[iGlobal];
}

/****************************************************************************************/
/************************************** Damping *****************************************/
/****************************************************************************************/
__global__ void dampCosineEdge(float *dev_p1, float *dev_p2) {

	int izGlobal = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // Global z-coordinate
	int ixGlobal = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // Global x-coordinate
	int iGlobal = dev_nz * ixGlobal + izGlobal; // 1D array index for the model on the global memory

	// Compute distance to the closest edge of model
	int distToEdge = min4(izGlobal-FAT, ixGlobal-FAT, dev_nz-izGlobal-1-FAT, dev_nx-ixGlobal-1-FAT);
	if (distToEdge < dev_minPad){

		// Compute damping coefficient
		float damp = dev_cosDampingCoeff[distToEdge];

		// Apply damping
		dev_p1[iGlobal] *= damp;
		dev_p2[iGlobal] *= damp;
	}
}

/****************************************************************************************/
/************************************** Scaling *****************************************/
/****************************************************************************************/
// Applying scaling to reflectivity for non extended imaging
// In that case, we can apply both scaling simultaneously
__global__ void scaleReflectivity(float *dev_model, float *dev_reflectivityScaleIn, float *dev_vel2Dtw2In){

	int izGlobal = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // Global z-coordinate
	int ixGlobal = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // Global x-coordinate
	int iGlobal = dev_nz * ixGlobal + izGlobal; // 1D array index for the model on the global memory
	dev_model[iGlobal] *= dev_vel2Dtw2In[iGlobal] * dev_reflectivityScaleIn[iGlobal];
}

// Scale the model perturbation by 2/v^3
// This is used for subsurface offset extension, where we can not apply both scaling simultaneously
__global__ void scaleReflectivityLinExt(float *dev_model, float *dev_reflectivityScaleIn){

	int iz = FAT + blockIdx.x * BLOCK_SIZE_EXT + threadIdx.x; // z-coordinate
	int ix = FAT + blockIdx.y * BLOCK_SIZE_EXT + threadIdx.y; // x-coordinate
	int iSpace = dev_nz * ix + iz;
	int iExt = blockIdx.z * BLOCK_SIZE_EXT + threadIdx.z; // Extended axis coordinate
	int iModel = iExt * dev_nz * dev_nx + iSpace; // 1D array index for the model on the global memory

	if (iExt < dev_nExt){
		dev_model[iModel] *= dev_reflectivityScaleIn[iSpace];
	}
}

// Scale the secondary source by dtw^2 * v^2
__global__ void scaleSecondarySourceFd(float *dev_timeSlice, float *dev_vel2Dtw2In){
	int iz = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // z-coordinate
	int ix = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // x-coordinate
	int iSpace = dev_nz * ix + iz;
	dev_timeSlice[iSpace] *= dev_vel2Dtw2In[iSpace];
}

// Apply both scalings (linearization + finite difference) to the extended reflectivity (only works for time-lags extension)
__global__ void scaleReflectivityExt(float *dev_model, float *dev_reflectivityScaleIn, float *dev_vel2Dtw2In){

	int iz = FAT + blockIdx.x * BLOCK_SIZE_EXT + threadIdx.x; // z-coordinate
	int ix = FAT + blockIdx.y * BLOCK_SIZE_EXT + threadIdx.y; // x-coordinate
	int iSpace = dev_nz * ix + iz;
	int iExt = blockIdx.z * BLOCK_SIZE_EXT + threadIdx.z; // Extended axis coordinate
	int iModel = iExt * dev_nz * dev_nx + iSpace; // 1D array index for the model on the global memory

	if (iExt < dev_nExt){
		dev_model[iModel] *= dev_reflectivityScaleIn[iSpace] * dev_vel2Dtw2In[iSpace];
	}
}

/****************************************************************************************/
/************************************** Imaging *****************************************/
/****************************************************************************************/
// Non-extended
__global__ void imagingFwdGpu(float *dev_model, float *dev_timeSlice, int its, float *dev_sourceWavefieldDts) {

	int izGlobal = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // Global z-coordinate
	int ixGlobal = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // Global x-coordinate
	int iGlobal = dev_nz * ixGlobal + izGlobal; // 1D array index for the model on the global memory
	int iGlobalWavefield = its * dev_nz * dev_nx + iGlobal;
	dev_timeSlice[iGlobal] = dev_model[iGlobal] * dev_sourceWavefieldDts[iGlobalWavefield];
}

__global__ void imagingAdjGpu(float *dev_model, float *dev_timeSlice, float *dev_srcWavefieldDts, int its){

	int izGlobal = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // Global z-coordinate
	int ixGlobal = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // Global x-coordinate
	int iGlobal = dev_nz * ixGlobal + izGlobal; // 1D array index for the model on the global memory
	int iGlobalWavefield = its * dev_nz * dev_nx + iGlobal;
	dev_model[iGlobal] += dev_srcWavefieldDts[iGlobalWavefield] * dev_timeSlice[iGlobal];
}

__global__ void imagingAdjTomoGpu(float *dev_wavefieldIn, float *dev_timeSliceOut, float *dev_extReflectivityIn, int its) {

	int izGlobal = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // Global z-coordinate
	int ixGlobal = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // Global x-coordinate
	int iGlobal = dev_nz * ixGlobal + izGlobal; // 1D array index for the model on the global memory
	int iGlobalWavefield = its * dev_nz * dev_nx + iGlobal;
	dev_timeSliceOut[iGlobal] = dev_extReflectivityIn[iGlobal] * dev_wavefieldIn[iGlobalWavefield];
}

// Time-lags
__global__ void imagingTimeFwdGpu(float *dev_model, float *dev_timeSlice, float *dev_srcWavefieldDts, int its, int iExtMin, int iExtMax){

	int iz = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // Global z-coordinate
	int ix = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // Global x-coordinate
	int iSpace = dev_nz * ix + iz; // 1D array index for the model on the global memory
	int iWavefield = its * dev_nz * dev_nx + iSpace; // Index for source wavefield at its

	for (int iExt = iExtMin; iExt < iExtMax; iExt++){

		int iModelExt = iExt * dev_nz * dev_nx + iSpace; // Compute index for extended model
		int iSrcWavefield = iWavefield - 2 * (iExt-dev_hExt) * dev_nz * dev_nx; // Compute index for source wavefield
		dev_timeSlice[iSpace] += dev_model[iModelExt] * dev_srcWavefieldDts[iSrcWavefield]; // Compute FWD imaging condition
	}
}

__global__ void imagingTimeAdjGpu(float *dev_model, float *dev_receiverTimeSlice, float *dev_srcWavefieldDts, int its, int iExtMin, int iExtMax){

	int iz = FAT + blockIdx.x * BLOCK_SIZE_EXT + threadIdx.x; // z-coordinate
	int ix = FAT + blockIdx.y * BLOCK_SIZE_EXT + threadIdx.y; // x-coordinate
	int iSpace = dev_nz * ix + iz; // 1D array index on spatial grid
	int iExt = iExtMin + blockIdx.z * BLOCK_SIZE_EXT + threadIdx.z; // Extended axis coordinate
	int iSrcWavefield = (its-2*(iExt-dev_hExt)) * dev_nz * dev_nx + iSpace; // Index for source wavefield at its
	int iModel = iExt * dev_nz * dev_nx + iSpace; // Extended model index

	if (iExt < iExtMax){
		dev_model[iModel] += dev_receiverTimeSlice[iSpace] * dev_srcWavefieldDts[iSrcWavefield]; // Try without +=
	}
}

__global__ void imagingWemvaTimeAdjGpu(float *dev_model, float *dev_scatteredTimeSlice, float *dev_recWavefield, int its, int iExtMin, int iExtMax){

	int iz = FAT + blockIdx.x * BLOCK_SIZE_EXT + threadIdx.x; // z-coordinate
	int ix = FAT + blockIdx.y * BLOCK_SIZE_EXT + threadIdx.y; // x-coordinate
	int iSpace = dev_nz * ix + iz; // 1D array index on spatial grid
	int iExt = iExtMin + blockIdx.z * BLOCK_SIZE_EXT + threadIdx.z; // Extended axis coordinate
	int iRecWavefield = (its+2*(iExt-dev_hExt)) * dev_nz * dev_nx + iSpace; // Index for receiver wavefield at its
	int iModel = iExt * dev_nz * dev_nx + iSpace; // Extended model index

	if (iExt < iExtMax){
		dev_model[iModel] += dev_scatteredTimeSlice[iSpace] * dev_recWavefield[iRecWavefield]; // Try without +=
	}
}

// Subsurface offsets
__global__ void imagingOffsetFwdGpu(float *dev_model, float *dev_timeSlice, float *dev_srcWavefieldDts, int its){

	int iz = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // z-coordinate on main grid
	int ix = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // x-coordinate on main grid
	int iSpace = dev_nz * ix + iz; // 1D array index for the model on the global memory

	int iExtMin=max2(-dev_hExt, ix+1+FAT+dev_hExt-dev_nx);
	int iExtMax=min2(ix-dev_hExt-FAT, dev_hExt)+1;

	for (int iExt=iExtMin; iExt<iExtMax; iExt++){

		int iModel = dev_nz * dev_nx * (iExt+dev_hExt) + dev_nz * (ix-iExt) + iz; // model(iz, ix-iOffset, iOffset+hOffset)
		int iSrcWavefield = dev_nz * dev_nx * its + (ix-2*iExt) * dev_nz + iz; // src(iz, ix-2*iOffset, its)
		dev_timeSlice[iSpace] += dev_model[iModel] * dev_srcWavefieldDts[iSrcWavefield];
	}
}

__global__ void imagingOffsetAdjGpu(float *dev_model, float *dev_timeSlice, float *dev_srcWavefieldDts, int its){

	long long iz = FAT + blockIdx.x * BLOCK_SIZE_EXT + threadIdx.x; // z-coordinate on main grid
	long long ix = FAT + dev_hExt + blockIdx.y * BLOCK_SIZE_EXT + threadIdx.y; // x-coordinate on main grid for the model where we evaluate the image
	long long iExt = blockIdx.z * BLOCK_SIZE_EXT + threadIdx.z; // offset coordinate (iOffset = 0, ..., dev_nOffset-1)

	if ( (ix < dev_nx-FAT-dev_hExt) && (iExt < dev_nExt) ){
		long long iExtShift=iExt-dev_hExt;
		long long iModel = dev_nz * dev_nx * iExt + dev_nz * ix + iz; // Model index
		long long iSrcWavefield = dev_nz * dev_nx * its + dev_nz * (ix-iExtShift) + iz; // Source wavefield index
		long long iRecWavefield = dev_nz * (ix+iExtShift) + iz; // Receiver wavefield index
		dev_model[iModel] += dev_timeSlice[iRecWavefield] * dev_srcWavefieldDts[iSrcWavefield];
	}
}

// Extended imaging condition for Wemva fwd
__global__ void imagingOffsetWemvaFwdGpu(float *dev_model, float *dev_timeSlice, float *dev_recWavefield, int its){

	int iz = FAT + blockIdx.x * BLOCK_SIZE_EXT + threadIdx.x; // z-coordinate on main grid
	int ix = FAT + dev_hExt + blockIdx.y * BLOCK_SIZE_EXT + threadIdx.y; // x-coordinate on main grid for the model where we evaluate the image
	int iExt = blockIdx.z * BLOCK_SIZE_EXT + threadIdx.z; // offset coordinate (iOffset = 0, ..., dev_nOffset-1)

	if ( (ix < dev_nx-FAT-dev_hExt) && (iExt < dev_nExt) ){
		int iExtShift=iExt-dev_hExt;
		int iModel = dev_nz * dev_nx * iExt + dev_nz * ix + iz; // Model index for extended image
		int iRecWavefield = dev_nz * dev_nx * its + dev_nz * (ix+iExtShift) + iz; // Receiver wavefield index
		int iScatWavefield = dev_nz * (ix-iExtShift) + iz; // Scattered wavefield index
		dev_model[iModel] += dev_timeSlice[iScatWavefield] * dev_recWavefield[iRecWavefield];
	}
}

__global__ void imagingOffsetWemvaScaleFwdGpu(float *dev_model, float *dev_timeSlice, float *dev_recWavefield, float *dev_vel2Dtw2In, int its){

	int iz = FAT + blockIdx.x * BLOCK_SIZE_EXT + threadIdx.x; // z-coordinate on main grid
	int ix = FAT + dev_hExt + blockIdx.y * BLOCK_SIZE_EXT + threadIdx.y; // x-coordinate on main grid for the model where we evaluate the image
	int iExt = blockIdx.z * BLOCK_SIZE_EXT + threadIdx.z; // offset coordinate (iOffset = 0, ..., dev_nOffset-1)

	if ( (ix < dev_nx-FAT-dev_hExt) && (iExt < dev_nExt) ){
		int iExtShift=iExt-dev_hExt;
		int iModel = dev_nz * dev_nx * iExt + dev_nz * ix + iz; // Model index for extended image
		int iRecWavefieldScale = (ix+iExtShift) * dev_nz + iz; // Index for vel2dtw2(iz, ix + ihx)
		int iRecWavefield = dev_nz * dev_nx * its + iRecWavefieldScale; // Receiver wavefield index
		int iScatWavefield = dev_nz * (ix-iExtShift) + iz; // Scattered wavefield index
		dev_model[iModel] += dev_timeSlice[iScatWavefield] * dev_recWavefield[iRecWavefield] * dev_vel2Dtw2In[iRecWavefieldScale];
	}
}

__global__ void imagingTimeTomoAdjGpu(float *dev_wavefieldIn, float *dev_timeSliceOut, float *dev_extReflectivityIn, int its, int iExtMin, int iExtMax) {

	int iz = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // Global z-coordinate
	int ix = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // Global x-coordinate
	int iSpace = dev_nz * ix + iz; // 1D array index for the model on the global memory
	int iWavefield = its * dev_nz * dev_nx + iSpace; // Index for source wavefield at its
	// Put pragma unroll statement for speed up
	for (int iExt = iExtMin; iExt < iExtMax; iExt++){

		int iModelExt = iExt * dev_nz * dev_nx + iSpace; // Compute index for extended model
		int iRecWavefield = iWavefield + 2 * (iExt-dev_hExt) * dev_nz * dev_nx; // Compute index for source wavefield
		dev_timeSliceOut[iSpace] += dev_extReflectivityIn[iModelExt] * dev_wavefieldIn[iRecWavefield];
	}
}

__global__ void imagingOffsetTomoAdjGpu(float *dev_wavefieldIn, float *dev_timeSliceOut, float *dev_extReflectivityIn, float *dev_vel2Dtw2In, int its) {

	int iz = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // z-coordinate on main grid
	int ix = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // x-coordinate on main grid
	int iSpace = dev_nz * ix + iz; // 1D array index for the model on the global memory

	int iExtMin=max2(-dev_hExt, FAT+dev_hExt-ix);
	int iExtMax=min2(dev_nx-1-FAT-ix-dev_hExt, dev_hExt)+1;

	for (int iExt=iExtMin; iExt<iExtMax; iExt++){
		int iModel = dev_nz * dev_nx * (iExt+dev_hExt) + dev_nz * (ix+iExt) + iz; // model(iz, ix-iOffset, iOffset+hOffset)
		int iRecWavefieldScale = (ix+2*iExt) * dev_nz + iz; //Index for vel2dtw2(iz, ix + 2*iOffset)
		int iRecWavefield = dev_nz * dev_nx * its + iRecWavefieldScale; // rec(iz, ix + 2*iOffset, its)
		dev_timeSliceOut[iSpace] += dev_extReflectivityIn[iModel] * dev_wavefieldIn[iRecWavefield] * dev_vel2Dtw2In[iRecWavefieldScale];
	}
}

__global__ void imagingOffsetTomoAdjNoFdScaleGpu(float *dev_wavefieldIn, float *dev_timeSliceOut, float *dev_extReflectivityIn, int its) {

	int iz = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // z-coordinate on main grid
	int ix = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // x-coordinate on main grid
	int iSpace = dev_nz * ix + iz; // 1D array index for the model on the global memory

	int iExtMin=max2(-dev_hExt, FAT+dev_hExt-ix);
	int iExtMax=min2(dev_nx-1-FAT-ix-dev_hExt, dev_hExt)+1;

	for (int iExt=iExtMin; iExt<iExtMax; iExt++){
		int iModel = dev_nz * dev_nx * (iExt+dev_hExt) + dev_nz * (ix+iExt) + iz; // model(iz, ix-iOffset, iOffset+hOffset)
		int iRecWavefield = dev_nz * dev_nx * its + (ix+2*iExt) * dev_nz + iz; // rec(iz, ix + 2*iOffset, its)
		dev_timeSliceOut[iSpace] += dev_extReflectivityIn[iModel] * dev_wavefieldIn[iRecWavefield];
	}
}

__global__ void imagingOffsetWemvaAdjGpu(float *dev_wavefieldIn, float *dev_timeSliceOut, float *dev_extReflectivityIn, int its) {

	int iz = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // z-coordinate on main grid
	int ix = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // x-coordinate on main grid
	int iSpace = dev_nz * ix + iz; // 1D array index for the model on the global memory

	int iExtMin=max2(-dev_hExt, FAT+dev_hExt-ix);
	int iExtMax=min2(dev_nx-1-FAT-ix, dev_hExt)+1;

	for (int iExt=iExtMin; iExt<iExtMax; iExt++){
		int iModel = dev_nz * dev_nx * (iExt+dev_hExt) + dev_nz * (ix-iExt) + iz; // model(iz, ix-iOffset, iOffset+hOffset)
		int iSrcWavefield = dev_nz * dev_nx * its + (ix-2*iExt) * dev_nz + iz; // src(iz, ix-2*iOffset, its)
		dev_timeSliceOut[iSpace] += dev_extReflectivityIn[iModel] * dev_wavefieldIn[iSrcWavefield];
	}
}

/****************************************************************************************/
/*********************************** Forward steppers ***********************************/
/****************************************************************************************/
/* Forward stepper (no damping) */
__global__ void stepFwdGpu(float *dev_o, float *dev_c, float *dev_n, float *dev_vel2Dtw2) {

	__shared__ float shared_c[BLOCK_SIZE+2*FAT][BLOCK_SIZE+2*FAT]; // Allocate shared memory
	int izGlobal = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // Global z-coordinate
	int ixGlobal = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // Global x-coordinate
	int izLocal = FAT + threadIdx.x; // z-coordinate on the shared grid
	int ixLocal = FAT + threadIdx.y; // x-coordinate on the shared grid
	int iGlobal = dev_nz * ixGlobal + izGlobal; // 1D array index for the model on the global memory

	// Copy current slice from global to shared memory
	// Each thread is going to perform this operation
	shared_c[ixLocal][izLocal] = dev_c[iGlobal];

	// Copy current slice from global to shared -- edges
	if (threadIdx.y < FAT) {
		shared_c[ixLocal-FAT][izLocal] = dev_c[iGlobal-dev_nz*FAT]; // Left side
		shared_c[ixLocal+BLOCK_SIZE][izLocal] = dev_c[iGlobal+dev_nz*BLOCK_SIZE] ; // Right side
	}
	if (threadIdx.x < FAT) {
		shared_c[ixLocal][izLocal-FAT] = dev_c[iGlobal-FAT]; // Up
		shared_c[ixLocal][izLocal+BLOCK_SIZE] = dev_c[iGlobal+BLOCK_SIZE]; // Down
	}
	__syncthreads(); // Synchronise all threads within each block
	// For a given block, we have now loaded the entire "block slice" plus the halos on both directions into the shared memory
	// We can now compute the Laplacian value at each point of the entire block slice

	dev_n[iGlobal] =  dev_vel2Dtw2[iGlobal] * ( dev_zCoeff[0] * shared_c[ixLocal][izLocal]
				   +  dev_zCoeff[1] * ( shared_c[ixLocal][izLocal-1] + shared_c[ixLocal][izLocal+1] )
				   +  dev_zCoeff[2] * ( shared_c[ixLocal][izLocal-2] + shared_c[ixLocal][izLocal+2] )
				   +  dev_zCoeff[3] * ( shared_c[ixLocal][izLocal-3] + shared_c[ixLocal][izLocal+3] )
				   +  dev_zCoeff[4] * ( shared_c[ixLocal][izLocal-4] + shared_c[ixLocal][izLocal+4] )
				   +  dev_zCoeff[5] * ( shared_c[ixLocal][izLocal-5] + shared_c[ixLocal][izLocal+5] )
				   +  dev_xCoeff[0] * shared_c[ixLocal][izLocal]
				   +  dev_xCoeff[1] * ( shared_c[ixLocal+1][izLocal] + shared_c[ixLocal-1][izLocal] )
				   +  dev_xCoeff[2] * ( shared_c[ixLocal+2][izLocal] + shared_c[ixLocal-2][izLocal] )
				   +  dev_xCoeff[3] * ( shared_c[ixLocal+3][izLocal] + shared_c[ixLocal-3][izLocal] )
				   +  dev_xCoeff[4] * ( shared_c[ixLocal+4][izLocal] + shared_c[ixLocal-4][izLocal] )
				   +  dev_xCoeff[5] * ( shared_c[ixLocal+5][izLocal] + shared_c[ixLocal-5][izLocal] ) )
				   +  shared_c[ixLocal][izLocal] + shared_c[ixLocal][izLocal] - dev_o[iGlobal];
}

/****************************************************************************************/
/*********************************** Adjoint steppers ***********************************/
/****************************************************************************************/

/* Adjoint stepper (no damping) */
__global__ void stepAdjGpu(float *dev_o, float *dev_c, float *dev_n, float *dev_vel2Dtw2) {

	__shared__ float shared_c[BLOCK_SIZE+2*FAT][BLOCK_SIZE+2*FAT]; // Allocate shared memory
	int izGlobal = FAT + blockIdx.x * BLOCK_SIZE + threadIdx.x; // Global z-coordinate
	int ixGlobal = FAT + blockIdx.y * BLOCK_SIZE + threadIdx.y; // Global x-coordinate
	int izLocal = FAT + threadIdx.x; // z-coordinate on the shared grid
	int ixLocal = FAT + threadIdx.y; // z-coordinate on the shared grid
	int iGlobal = dev_nz * ixGlobal + izGlobal; // 1D array index for the model on the global memory

	// Copy current slice from global memory to shared memory
	shared_c[ixLocal][izLocal] = dev_c[iGlobal] * dev_vel2Dtw2[iGlobal];

	// Copy current slice from global memory to shared -- edges ("halo")
	if (threadIdx.y < FAT) {
		shared_c[ixLocal-FAT][izLocal] = dev_c[iGlobal-dev_nz*FAT] * dev_vel2Dtw2[iGlobal-dev_nz*FAT]; // Left side
		shared_c[ixLocal+BLOCK_SIZE][izLocal] = dev_c[iGlobal+dev_nz*BLOCK_SIZE] * dev_vel2Dtw2[iGlobal+dev_nz*BLOCK_SIZE]; // Right side
	}
	if (threadIdx.x < FAT) {
		shared_c[ixLocal][izLocal-FAT] = dev_c[iGlobal-FAT] * dev_vel2Dtw2[iGlobal-FAT]; // Up
		shared_c[ixLocal][izLocal+BLOCK_SIZE] = dev_c[iGlobal+BLOCK_SIZE] * dev_vel2Dtw2[iGlobal+BLOCK_SIZE]; // Down
	}
	__syncthreads(); // Synchronise all threads within each block

	dev_o[iGlobal] =  ( dev_zCoeff[0] * shared_c[ixLocal][izLocal]
				   +  dev_zCoeff[1] * ( shared_c[ixLocal][izLocal-1] + shared_c[ixLocal][izLocal+1] )
				   +  dev_zCoeff[2] * ( shared_c[ixLocal][izLocal-2] + shared_c[ixLocal][izLocal+2] )
				   +  dev_zCoeff[3] * ( shared_c[ixLocal][izLocal-3] + shared_c[ixLocal][izLocal+3] )
				   +  dev_zCoeff[4] * ( shared_c[ixLocal][izLocal-4] + shared_c[ixLocal][izLocal+4] )
				   +  dev_zCoeff[5] * ( shared_c[ixLocal][izLocal-5] + shared_c[ixLocal][izLocal+5] )
				   +  dev_xCoeff[0] * shared_c[ixLocal][izLocal]
				   +  dev_xCoeff[1] * ( shared_c[ixLocal+1][izLocal] + shared_c[ixLocal-1][izLocal] )
				   +  dev_xCoeff[2] * ( shared_c[ixLocal+2][izLocal] + shared_c[ixLocal-2][izLocal] )
				   +  dev_xCoeff[3] * ( shared_c[ixLocal+3][izLocal] + shared_c[ixLocal-3][izLocal] )
				   +  dev_xCoeff[4] * ( shared_c[ixLocal+4][izLocal] + shared_c[ixLocal-4][izLocal] )
				   +  dev_xCoeff[5] * ( shared_c[ixLocal+5][izLocal] + shared_c[ixLocal-5][izLocal] ) )
				   +  dev_c[iGlobal] + dev_c[iGlobal] - dev_n[iGlobal];
}
