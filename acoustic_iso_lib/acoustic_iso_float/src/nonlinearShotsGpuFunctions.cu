#include "hip/hip_runtime.h"
#include <cstring>
#include <iostream>
#include "nonlinearShotsGpuFunctions.h"
#include "varDeclare.h"
#include "kernelsGpu.cu"
#include "cudaErrors.cu"
#include <vector>
#include <algorithm>
#include <math.h>
#include <omp.h>
#include <ctime>
#include <stdio.h>
#include <assert.h>

/****************************************************************************************/
/******************************* Set GPU propagation parameters *************************/
/****************************************************************************************/
bool getGpuInfo(int nGpu, int info, int deviceNumberInfo){

	int nDevice, driver;
	hipGetDeviceCount(&nDevice);

	if (info == 1){

		std::cout << " " << std::endl;
		std::cout << "-------------------------------------------------------------------" << std::endl;
		std::cout << "---------------------------- INFO FOR GPU# " << deviceNumberInfo << " ----------------------" << std::endl;
		std::cout << "-------------------------------------------------------------------" << std::endl;

		// Number of devices
		std::cout << "Number of requested GPUs: " << nGpu << std::endl;
		std::cout << "Number of available GPUs: " << nDevice << std::endl;

		// Driver version
		std::cout << "Cuda driver version: " << hipDriverGetVersion(&driver) << std::endl; // Driver

		// Get properties
		hipDeviceProp_t dprop;
		hipGetDeviceProperties(&dprop,deviceNumberInfo);

		// Display
		std::cout << "Name: " << dprop.name << std::endl;
		std::cout << "Total global memory: " << dprop.totalGlobalMem/(1024*1024*1024) << " [GB] " << std::endl;
		std::cout << "Shared memory per block: " << dprop.sharedMemPerBlock/1024 << " [kB]" << std::endl;
		std::cout << "Number of register per block: " << dprop.regsPerBlock << std::endl;
		std::cout << "Warp size: " << dprop.warpSize << " [threads]" << std::endl;
		std::cout << "Maximum pitch allowed for memory copies in bytes: " << dprop.memPitch/(1024*1024*1024) << " [GB]" << std::endl;
		std::cout << "Maximum threads per block: " << dprop.maxThreadsPerBlock << std::endl;
		std::cout << "Maximum block dimensions: " << "(" << dprop.maxThreadsDim[0] << ", " << dprop.maxThreadsDim[1] << ", " << dprop.maxThreadsDim[2] << ")" << std::endl;
		std::cout << "Maximum grid dimensions: " << "(" << dprop.maxGridSize[0] << ", " << dprop.maxGridSize[1] << ", " << dprop.maxGridSize[2] << ")" << std::endl;
		std::cout << "Total constant memory: " << dprop.totalConstMem/1024 << " [kB]" << std::endl;
		std::cout << "Number of streaming multiprocessors on device: " << dprop.multiProcessorCount << std::endl;
		if (dprop.deviceOverlap == 1) {std::cout << "Device can simultaneously perform a hipMemcpy() and kernel execution" << std::endl;}
		if (dprop.deviceOverlap != 1) {std::cout << "Device cannot simultaneously perform a hipMemcpy() and kernel execution" << std::endl;}
		if (dprop.canMapHostMemory == 1) { std::cout << "Device can map host memory" << std::endl; }
		if (dprop.canMapHostMemory != 1) { std::cout << "Device cannot map host memory" << std::endl; }
		if (dprop.concurrentKernels == 1) {std::cout << "Device can support concurrent kernel" << std::endl;}
		if (dprop.concurrentKernels != 1) {std::cout << "Device cannot support concurrent kernel execution" << std::endl;}

		std::cout << "-------------------------------------------------------------------" << std::endl;
		std::cout << " " << std::endl;
	}

  	if (nGpu<nDevice+1) {return true;}
  	else {std::cout << "Number of requested GPU greater than available GPUs" << std::endl; return false;}
}
void initNonlinearGpu(float dz, float dx, int nz, int nx, int nts, float dts, int sub, int minPad, int blockSize, float alphaCos, int nGpu, int iGpu){

	// Set GPU
	hipSetDevice(iGpu);

	// Host variables
	host_nz = nz;
	host_nx = nx;
	host_nts = nts;
	host_sub = sub;
	host_ntw = (nts - 1) * sub + 1;

	/**************************** ALLOCATE ARRAYS OF ARRAYS *****************************/
	// Only one GPU will perform the following
	if (iGpu == 0) {

		// Time slices for FD stepping
		dev_p0 = new float*[nGpu];
		dev_p1 = new float*[nGpu];
		dev_temp1 = new float*[nGpu];

		// Data and model
		dev_modelRegDtw = new float*[nGpu];
		dev_dataRegDts = new float*[nGpu];

		// Source and receivers
		dev_sourcesPositionReg = new int*[nGpu];
		dev_receiversPositionReg = new int*[nGpu];

		// Scaled velocity
		dev_vel2Dtw2 = new float*[nGpu];

	}

	/**************************** COMPUTE LAPLACIAN COEFFICIENTS ************************/
	float zCoeff[COEFF_SIZE];
	float xCoeff[COEFF_SIZE];

	zCoeff[0] = -2.927222222 / (dz * dz);
  	zCoeff[1] = 1.666666667 / (dz * dz);
  	zCoeff[2] = -0.238095238 / (dz * dz);
  	zCoeff[3] = 0.039682539 / (dz * dz);
  	zCoeff[4] = -0.004960317 / (dz * dz);
  	zCoeff[5] = 0.000317460 / (dz * dz);

  	xCoeff[0] = -2.927222222 / (dx * dx);
  	xCoeff[1] = 1.666666667 / (dx * dx);
  	xCoeff[2] = -0.238095238 / (dx * dx);
  	xCoeff[3] = 0.039682539 / (dx * dx);
  	xCoeff[4] = -0.004960317 / (dx * dx);
  	xCoeff[5] = 0.000317460 / (dx * dx);

	/**************************** COMPUTE TIME-INTERPOLATION FILTER *********************/
	// Time interpolation filter length/half length
	int hInterpFilter = host_sub + 1;
	int nInterpFilter = 2 * hInterpFilter;

	// Check the subsampling coefficient is smaller than the maximum allowed
	if (sub>=SUB_MAX){
		std::cout << "**** ERROR: Subsampling parameter too high ****" << std::endl;
		assert (1==2);
	}

	// Allocate and fill interpolation filter
	float interpFilter[nInterpFilter];
	for (int iFilter = 0; iFilter < hInterpFilter; iFilter++){
		interpFilter[iFilter] = 1.0 - 1.0 * iFilter/host_sub;
		interpFilter[iFilter+hInterpFilter] = 1.0 - interpFilter[iFilter];
		interpFilter[iFilter] = interpFilter[iFilter] * (1.0 / sqrt(float(host_ntw)/float(host_nts)));
		interpFilter[iFilter+hInterpFilter] = interpFilter[iFilter+hInterpFilter] * (1.0 / sqrt(float(host_ntw)/float(host_nts)));
	}

	/************************* COMPUTE COSINE DAMPING COEFFICIENTS **********************/
	if (minPad>=PAD_MAX){
		std::cout << "**** ERROR: Padding value is too high ****" << std::endl;
		assert (1==2);
	}
	float cosDampingCoeff[minPad];

	// Cosine padding
	for (int iFilter=FAT; iFilter<FAT+minPad; iFilter++){
		float arg = M_PI / (1.0 * minPad) * 1.0 * (minPad-iFilter+FAT);
		arg = alphaCos + (1.0-alphaCos) * cos(arg);
		cosDampingCoeff[iFilter-FAT] = arg;
	}

	// Check that the block size is consistent between parfile and "varDeclare.h"
	if (blockSize != BLOCK_SIZE) {
		std::cout << "**** ERROR: Block size for time stepper is not consistent with parfile ****" << std::endl;
		assert (1==2);
	}

	/**************************** COPY TO CONSTANT MEMORY *******************************/
	// Laplacian coefficients
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_zCoeff), zCoeff, COEFF_SIZE*sizeof(float), 0, hipMemcpyHostToDevice)); // Copy Laplacian coefficients to device
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_xCoeff), xCoeff, COEFF_SIZE*sizeof(float), 0, hipMemcpyHostToDevice));

	// Time interpolation filter
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nInterpFilter), &nInterpFilter, sizeof(int), 0, hipMemcpyHostToDevice)); // Filter length
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_hInterpFilter), &hInterpFilter, sizeof(int), 0, hipMemcpyHostToDevice)); // Filter half-length
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_interpFilter), interpFilter, nInterpFilter*sizeof(float), 0, hipMemcpyHostToDevice)); // Filter

	// Cosine damping parameters
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_cosDampingCoeff), &cosDampingCoeff, minPad*sizeof(float), 0, hipMemcpyHostToDevice)); // Array for damping
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_alphaCos), &alphaCos, sizeof(float), 0, hipMemcpyHostToDevice)); // Coefficient in the damping formula
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_minPad), &minPad, sizeof(int), 0, hipMemcpyHostToDevice)); // min (zPadMinus, zPadPlus, xPadMinus, xPadPlus)

	// FD parameters
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nz), &nz, sizeof(int), 0, hipMemcpyHostToDevice)); // Copy model size to device
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nx), &nx, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nts), &nts, sizeof(int), 0, hipMemcpyHostToDevice)); // Copy number of coarse time parameters to device
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_sub), &sub, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_ntw), &host_ntw, sizeof(int), 0, hipMemcpyHostToDevice)); // Copy number of coarse time parameters to device

}
void allocateNonlinearGpu(float *vel2Dtw2, int iGpu){

	// Get GPU number
	hipSetDevice(iGpu);

	// Scaled velocity
	cuda_call(hipMalloc((void**) &dev_vel2Dtw2[iGpu], host_nz*host_nx*sizeof(float))); // Allocate scaled velocity model on device
	cuda_call(hipMemcpy(dev_vel2Dtw2[iGpu], vel2Dtw2, host_nz*host_nx*sizeof(float), hipMemcpyHostToDevice));

	// Allocate time slices on device
	cuda_call(hipMalloc((void**) &dev_p0[iGpu], host_nz*host_nx*sizeof(float))); // Allocate time slices on device (for the stepper)
	cuda_call(hipMalloc((void**) &dev_p1[iGpu], host_nz*host_nx*sizeof(float)));

}
void deallocateNonlinearGpu(int iGpu){
		hipSetDevice(iGpu); // Set device number on GPU cluster
    	cuda_call(hipFree(dev_vel2Dtw2[iGpu])); // Deallocate scaled velocity
		cuda_call(hipFree(dev_p0[iGpu]));
    	cuda_call(hipFree(dev_p1[iGpu]));
}

/****************************************************************************************/
/******************************* Nonlinear forward propagation **************************/
/****************************************************************************************/
void propShotsFwdGpu(float *modelRegDtw, float *dataRegDts, int *sourcesPositionReg, int nSourcesReg, int *receiversPositionReg, int nReceiversReg, float *wavefieldDts, int iGpu) {

	// Set device number on GPU cluster
	hipSetDevice(iGpu);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(int)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(int), hipMemcpyHostToDevice));

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(int)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(int), hipMemcpyHostToDevice));

	// Model
  	cuda_call(hipMalloc((void**) &dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(float))); // Allocate input on device
	cuda_call(hipMemcpy(dev_modelRegDtw[iGpu], modelRegDtw, nSourcesReg*host_ntw*sizeof(float), hipMemcpyHostToDevice)); // Copy input signals on device

	// Data
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(float))); // Allocate output on device
  	cuda_call(hipMemset(dev_dataRegDts[iGpu], 0, nReceiversReg*host_nts*sizeof(float))); // Initialize output on device

	// Time slices
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(float)));
	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(float)));

	// Laplacian grid and blocks
	int nblockx = (host_nz-2*FAT) / BLOCK_SIZE;
	int nblocky = (host_nx-2*FAT) / BLOCK_SIZE;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	// Extraction grid size
	int nblockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;

	// Timer
	// std::clock_t start;
	// float duration;
	// start = std::clock();

	// Start propagation
	for (int its = 0; its < host_nts-1; its++){

		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Compute fine time-step index
			int itw = its * host_sub + it2;

			// Step forward
			kernel_exec(stepFwdGpu<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject source
			kernel_exec(injectSource<<<1, nSourcesReg>>>(dev_modelRegDtw[iGpu], dev_p0[iGpu], itw-1, dev_sourcesPositionReg[iGpu]));

			// Damp wavefields
			kernel_exec(dampCosineEdge<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Extract and interpolate data
			kernel_exec(recordInterpData<<<nblockData, BLOCK_SIZE_DATA>>>(dev_p0[iGpu], dev_dataRegDts[iGpu], its, it2, dev_receiversPositionReg[iGpu]));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}
	}

	// duration = (std::clock() - start) / (float) CLOCKS_PER_SEC;
	// std::cout << "duration: " << duration << std::endl;

	// Copy data back to host
	cuda_call(hipMemcpy(dataRegDts, dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(float), hipMemcpyDeviceToHost));

	// Deallocate all slices
    cuda_call(hipFree(dev_modelRegDtw[iGpu]));
    cuda_call(hipFree(dev_dataRegDts[iGpu]));
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));

}
void propShotsFwdGpuWavefield(float *modelRegDtw, float *dataRegDts, int *sourcesPositionReg, int nSourcesReg, int *receiversPositionReg, int nReceiversReg, float *wavefieldDts, int iGpu) {

	// Set device number on GPU cluster
	hipSetDevice(iGpu);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(int)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(int), hipMemcpyHostToDevice));

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(int)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(int), hipMemcpyHostToDevice));

	// Model
  	cuda_call(hipMalloc((void**) &dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(float))); // Allocate input on device
	cuda_call(hipMemcpy(dev_modelRegDtw[iGpu], modelRegDtw, nSourcesReg*host_ntw*sizeof(float), hipMemcpyHostToDevice)); // Copy input signals on device

	// Data
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(float))); // Allocate output on device
  	cuda_call(hipMemset(dev_dataRegDts[iGpu], 0, nReceiversReg*host_nts*sizeof(float))); // Initialize output on device

 	// Wavefield
	cuda_call(hipMalloc((void**) &dev_wavefieldDts, host_nz*host_nx*host_nts*sizeof(float))); // Allocate on device
	cuda_call(hipMemset(dev_wavefieldDts, 0, host_nz*host_nx*host_nts*sizeof(float))); // Initialize wavefield on device

	// Time slices
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(float))); // Initialize time slices on device
	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(float)));

	// Laplacian grid and blocks
	int nblockx = (host_nz-2*FAT) / BLOCK_SIZE;
	int nblocky = (host_nx-2*FAT) / BLOCK_SIZE;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	// Extraction grid size
	int nBlockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;

	// Start propagation
	for (int its = 0; its < host_nts-1; its++){

		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Compute fine time-step index
			int itw = its * host_sub + it2;

			// Step forward
			kernel_exec(stepFwdGpu<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject source
			kernel_exec(injectSource<<<1, nSourcesReg>>>(dev_modelRegDtw[iGpu], dev_p0[iGpu], itw-1, dev_sourcesPositionReg[iGpu]));

			// Damp wavefields
			kernel_exec(dampCosineEdge<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Extract wavefield
			kernel_exec(interpWavefield<<<dimGrid, dimBlock>>>(dev_wavefieldDts, dev_p0[iGpu], its, it2));

			// Extract and interpolate data
			kernel_exec(recordInterpData<<<nBlockData, BLOCK_SIZE_DATA>>>(dev_p0[iGpu], dev_dataRegDts[iGpu], its, it2, dev_receiversPositionReg[iGpu]));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}
	}

	// Copy data back to host
	cuda_call(hipMemcpy(dataRegDts, dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(float), hipMemcpyDeviceToHost));

	// Copy wavefield back to host
	cuda_call(hipMemcpy(wavefieldDts, dev_wavefieldDts, host_nz*host_nx*host_nts*sizeof(float), hipMemcpyDeviceToHost));

	// Deallocate all slices
    cuda_call(hipFree(dev_modelRegDtw[iGpu]));
    cuda_call(hipFree(dev_dataRegDts[iGpu]));
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));
	cuda_call(hipFree(dev_wavefieldDts));

}

/****************************************************************************************/
/******************************* Nonlinear adjoint propagation **************************/
/****************************************************************************************/
void propShotsAdjGpu(float *modelRegDtw, float *dataRegDts, int *sourcesPositionReg, int nSourcesReg, int *receiversPositionReg, int nReceiversReg, float *wavefieldDts, int iGpu) {

	// Set device number on GPU cluster
	hipSetDevice(iGpu);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(int)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(int), hipMemcpyHostToDevice));

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(int)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(int), hipMemcpyHostToDevice));

	// Model
  	cuda_call(hipMalloc((void**) &dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(float))); // Allocate model on device
  	cuda_call(hipMemset(dev_modelRegDtw[iGpu], 0, nSourcesReg*host_ntw*sizeof(float))); // Initialize model on device

	// Data
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(float))); // Allocate data on device
	cuda_call(hipMemcpy(dev_dataRegDts[iGpu], dataRegDts, nReceiversReg*host_nts*sizeof(float), hipMemcpyHostToDevice)); // Copy data on device

	// Initialize time slices on device
	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(float)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(float)));

	// Grid and block dimensions for stepper
	int nblockx = (host_nz-2*FAT) / BLOCK_SIZE;
	int nblocky = (host_nx-2*FAT) / BLOCK_SIZE;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	// Grid and block dimensions for data injection
	int nblockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;

	// Start propagation
	for (int its = host_nts-2; its > -1; its--){

		for (int it2 = host_sub-1; it2 > -1; it2--){

			// Compute fine time-step index
			int itw = its * host_sub + it2;

			// Step forward in time
			kernel_exec(stepAdjGpu<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject data
			kernel_exec(interpInjectData<<<nblockData, BLOCK_SIZE_DATA>>>(dev_dataRegDts[iGpu], dev_p0[iGpu], its, it2, dev_receiversPositionReg[iGpu]));

			// Damp wavefield
			kernel_exec(dampCosineEdge<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Extract model
			kernel_exec(recordSource<<<1, nSourcesReg>>>(dev_p0[iGpu], dev_modelRegDtw[iGpu], itw, dev_sourcesPositionReg[iGpu]));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;
		}
	}

	// Copy data back to host
	cuda_call(hipMemcpy(modelRegDtw, dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(float), hipMemcpyDeviceToHost));

	// Deallocate all slices
    cuda_call(hipFree(dev_modelRegDtw[iGpu]));
    cuda_call(hipFree(dev_dataRegDts[iGpu]));
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));

}
void propShotsAdjGpuWavefield(float *modelRegDtw, float *dataRegDts, int *sourcesPositionReg, int nSourcesReg, int *receiversPositionReg, int nReceiversReg, float *wavefieldDts, int iGpu) {

	// Set device number on GPU cluster
	hipSetDevice(iGpu);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(int)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(int), hipMemcpyHostToDevice));

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(int)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(int), hipMemcpyHostToDevice));

	// Model
  	cuda_call(hipMalloc((void**) &dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(float))); // Allocate model on device
  	cuda_call(hipMemset(dev_modelRegDtw[iGpu], 0, nSourcesReg*host_ntw*sizeof(float))); // Initialize model on device

	// Data
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(float))); // Allocate data on device
	cuda_call(hipMemcpy(dev_dataRegDts[iGpu], dataRegDts, nReceiversReg*host_nts*sizeof(float), hipMemcpyHostToDevice)); // Copy data on device

 	// Wavefield
	cuda_call(hipMalloc((void**) &dev_wavefieldDts, host_nz*host_nx*host_nts*sizeof(float))); // Allocate on device
	cuda_call(hipMemset(dev_wavefieldDts, 0, host_nz*host_nx*host_nts*sizeof(float))); // Initialize wavefield on device

	// Initialize time slices on device
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(float)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(float)));

	// Grid and block dimensions for stepper
	int nblockx = (host_nz-2*FAT) / BLOCK_SIZE;
	int nblocky = (host_nx-2*FAT) / BLOCK_SIZE;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	// Grid and block dimensions for data injection
	int nblockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;

	// Start propagation
	for (int its = host_nts-2; its > -1; its--){

		for (int it2 = host_sub-1; it2 > -1; it2--){

			// Compute fine time-step index
			int itw = its * host_sub + it2;

			// Step forward in time
			kernel_exec(stepAdjGpu<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject data
			kernel_exec(interpInjectData<<<nblockData, BLOCK_SIZE_DATA>>>(dev_dataRegDts[iGpu], dev_p0[iGpu], its, it2, dev_receiversPositionReg[iGpu]));

			// Damp wavefield
			kernel_exec(dampCosineEdge<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Interpolate and save wavefield on device (the wavefield is not scaled)
			kernel_exec(interpWavefield<<<dimGrid, dimBlock>>>(dev_wavefieldDts, dev_p0[iGpu], its, it2));

			// Extract model
			kernel_exec(recordSource<<<1, nSourcesReg>>>(dev_p0[iGpu], dev_modelRegDtw[iGpu], itw, dev_sourcesPositionReg[iGpu]));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;
		}
	}

	// Copy data back to host
	cuda_call(hipMemcpy(modelRegDtw, dev_modelRegDtw[iGpu], nSourcesReg*host_ntw*sizeof(float), hipMemcpyDeviceToHost));

	// Copy wavefield back to host
	cuda_call(hipMemcpy(wavefieldDts, dev_wavefieldDts, host_nz*host_nx*host_nts*sizeof(float), hipMemcpyDeviceToHost));

	// Deallocate all slices
    cuda_call(hipFree(dev_modelRegDtw[iGpu]));
    cuda_call(hipFree(dev_dataRegDts[iGpu]));
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));
	cuda_call(hipFree(dev_wavefieldDts));

}
