#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <math.h>
#include "tomoExtShotsGpuFunctions.h"
#include "kernelsGpu.cu"
#include "cudaErrors.cu"
#include "varDeclare.h"
#include <ctime>
#include <stdio.h>
#include <assert.h>

/****************************************************************************************/
/************************ Declaration of auxiliary functions ****************************/
/****************************************************************************************/
// Note: The implementations of these auxiliary functions are done at the bottom of the file

void computeTomoSrcWfldDt2(double *dev_sourcesIn, double *dev_wavefieldOut, int *dev_sourcesPositionsRegIn, int nSourcesRegIn, dim3 dimGridIn, dim3 dimBlockIn, int iGpu);
void computeTomoRecWfld(double *dev_dataIn, double *dev_wavefieldOut, int *dev_receiversPositionsRegIn, dim3 dimGridIn, dim3 dimBlockIn, int nBlockDataIn, int iGpu);

void computeTomoLeg1TimeFwd(double *dev_modelIn, double *dev_tomoSrcWavefieldDt2In, double *dev_dataOut, double *dev_wavefield1Out, double *dev_wavefield2Out, double *dev_extReflectivityIn, int *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int nBlockDataIn, int iGpu, int saveWavefield);
void computeTomoLeg2TimeFwd(double *dev_modelIn, double *dev_tomoSrcWavefieldDt2In, double *dev_dataOut, double *dev_wavefield1Out, double *dev_wavefield2Out, double *dev_extReflectivityIn, int *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int nBlockDataIn, int iGpu, int saveWavefield);

void computeTomoLeg1OffsetFwd(double *dev_modelIn, double *dev_tomoSrcWavefieldDt2In, double *dev_dataOut, double *dev_wavefield1Out, double *dev_wavefield2Out, double *dev_extReflectivityIn, int *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int nBlockDataIn, int iGpu, int saveWavefield);
void computeTomoLeg2OffsetFwd(double *dev_modelIn, double *dev_tomoSrcWavefieldDt2In, double *dev_dataOut, double *dev_wavefield1Out, double *dev_wavefield2Out, double *dev_extReflectivityIn, int *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int nBlockDataIn, int iGpu, int saveWavefield);

void computeTomoLeg1TimeAdj(double *dev_tomoSrcWavefieldDt2In, double *dev_tomoRecWavefieldIn, double *dev_modelTomoOut, double *dev_wavefield1Out, double *dev_extReflectivityIn, dim3 dimGridIn, dim3 dimBlockIn, int iGpu, int saveWavefield);
void computeTomoLeg2TimeAdj(double *dev_tomoSrcWavefieldDt2In, double *dev_tomoRecWavefieldIn, double *dev_modelTomoOut, double *dev_wavefield1Out, double *dev_extReflectivityIn, dim3 dimGridIn, dim3 dimBlockIn, int iGpu, int saveWavefield);

void computeTomoLeg1OffsetAdj(double *dev_tomoSrcWavefieldDt2In, double *dev_tomoRecWavefieldIn, double *dev_modelTomoOut, double *dev_wavefield1Out, double *dev_extReflectivityIn, dim3 dimGridIn, dim3 dimBlockIn, int iGpu, int saveWavefield);
void computeTomoLeg2OffsetAdj(double *dev_tomoSrcWavefieldDt2In, double *dev_tomoRecWavefieldIn, double *dev_modelTomoOut, double *dev_wavefield1Out, double *dev_extReflectivityIn, dim3 dimGridIn, dim3 dimBlockIn, int iGpu, int saveWavefield);

/****************************************************************************************/
/******************************* Set GPU propagation parameters *************************/
/****************************************************************************************/
// Display info on GPU
bool getGpuInfo(int nGpu, int info, int deviceNumberInfo){

	int nDevice, driver;
	hipGetDeviceCount(&nDevice);

	if (info == 1){

		std::cout << " " << std::endl;
		std::cout << "-------------------------------------------------------------------" << std::endl;
		std::cout << "---------------------------- INFO FOR GPU# " << deviceNumberInfo << " ----------------------" << std::endl;
		std::cout << "-------------------------------------------------------------------" << std::endl;

		// Number of devices
		std::cout << "Number of requested GPUs: " << nGpu << std::endl;
		std::cout << "Number of available GPUs: " << nDevice << std::endl;

		// Driver version
		hipDriverGetVersion(&driver);
		std::cout << "Cuda driver version: " << driver << std::endl; // Driver

		// Get properties
		hipDeviceProp_t dprop;
		hipGetDeviceProperties(&dprop,deviceNumberInfo);

		// Display
		std::cout << "Name: " << dprop.name << std::endl;
		std::cout << "Total global memory: " << dprop.totalGlobalMem/(1024*1024*1024) << " [GB] " << std::endl;
		std::cout << "Shared memory per block: " << dprop.sharedMemPerBlock/1024 << " [kB]" << std::endl;
		std::cout << "Number of register per block: " << dprop.regsPerBlock << std::endl;
		std::cout << "Warp size: " << dprop.warpSize << " [threads]" << std::endl;
		std::cout << "Maximum pitch allowed for memory copies in bytes: " << dprop.memPitch/(1024*1024*1024) << " [GB]" << std::endl;
		std::cout << "Maximum threads per block: " << dprop.maxThreadsPerBlock << std::endl;
		std::cout << "Maximum block dimensions: " << "(" << dprop.maxThreadsDim[0] << ", " << dprop.maxThreadsDim[1] << ", " << dprop.maxThreadsDim[2] << ")" << std::endl;
		std::cout << "Maximum grid dimensions: " << "(" << dprop.maxGridSize[0] << ", " << dprop.maxGridSize[1] << ", " << dprop.maxGridSize[2] << ")" << std::endl;
		std::cout << "Total constant memory: " << dprop.totalConstMem/1024 << " [kB]" << std::endl;
		std::cout << "Number of streaming multiprocessors on device: " << dprop.multiProcessorCount << std::endl;
		if (dprop.deviceOverlap == 1) {std::cout << "Device can simultaneously perform a hipMemcpy() and kernel execution" << std::endl;}
		if (dprop.deviceOverlap != 1) {std::cout << "Device cannot simultaneously perform a hipMemcpy() and kernel execution" << std::endl;}
		if (dprop.canMapHostMemory == 1) { std::cout << "Device can map host memory" << std::endl; }
		if (dprop.canMapHostMemory != 1) { std::cout << "Device cannot map host memory" << std::endl; }
		if (dprop.concurrentKernels == 1) {std::cout << "Device can support concurrent kernel" << std::endl;}
		if (dprop.concurrentKernels != 1) {std::cout << "Device cannot support concurrent kernel execution" << std::endl;}

		std::cout << "-------------------------------------------------------------------" << std::endl;
		std::cout << " " << std::endl;
	}

  	if (nGpu<nDevice+1) {return true;}
  	else {std::cout << "Number of requested GPU greater than available GPUs" << std::endl; return false;}
}

// Initialize GPU
void initTomoExtGpu(double dz, double dx, int nz, int nx, int nts, double dts, int sub, int minPad, int blockSize, double alphaCos, int nExt, int leg1, int leg2, int nGpu, int iGpu){

	// Set GPU number
	hipSetDevice(iGpu);

	host_nz = nz;
	host_nx = nx;
	host_dz = dz;
	host_dx = dx;
	host_nExt = nExt;
	host_hExt = (nExt-1)/2;
	host_nts = nts;
	host_dts = dts;
	host_sub = sub;
	host_ntw = (nts - 1) * sub + 1;
	host_leg1 = leg1;
	host_leg2 = leg2;

	// Coefficients for second-order time derivative
	host_cSide = 1.0 / (host_dts*host_dts);
	host_cCenter = -2.0 / (host_dts*host_dts);
	// host_cSide = 0.0;
	// host_cCenter = 1.0;

	/**************************** ALLOCATE ARRAYS OF ARRAYS *****************************/
	// Only one GPU will perform the following
	if (iGpu == 0) {

		// Time slices for FD stepping
		dev_p0 = new double*[nGpu];
		dev_p1 = new double*[nGpu];
		dev_temp1 = new double*[nGpu];

		// Time slices for FD stepping at coarse grid
		dev_ssLeft = new double*[nGpu];
		dev_ssRight = new double*[nGpu];
		dev_ssTemp1 = new double*[nGpu];

		// Time slices for FD stepping at coarse grid
		dev_scatLeft = new double*[nGpu];
		dev_scatRight = new double*[nGpu];
		dev_scatTemp1 = new double*[nGpu];

		// Time slices for FD stepping at coarse grid before second order time derivative
		dev_ss0 = new double*[nGpu];
		dev_ss1 = new double*[nGpu];
		dev_ss2 = new double*[nGpu];
		dev_ssTemp2 = new double*[nGpu];

		// Data
		dev_dataRegDts = new double*[nGpu];

		// Source and receivers
		dev_sourcesPositionReg = new int*[nGpu];
		dev_receiversPositionReg = new int*[nGpu];

		// Sources signal
		dev_sourcesSignals = new double*[nGpu];

		// Scaled velocity
		dev_vel2Dtw2 = new double*[nGpu];

		// Reflectivity scaling
		dev_reflectivityScale = new double*[nGpu];

		// Background perturbation ("model" for tomo)
		dev_modelTomo = new double*[nGpu];

		// Extended reflectivity for tomo
		dev_extReflectivity = new double*[nGpu];

		// Source and secondary wavefields
		dev_tomoSrcWavefieldDt2 = new double*[nGpu];
		dev_tomoSecWavefield1 = new double*[nGpu];
		dev_tomoSecWavefield2 = new double*[nGpu];

	}

	/**************************** COMPUTE LAPLACIAN COEFFICIENTS ************************/
	double zCoeff[COEFF_SIZE];
	double xCoeff[COEFF_SIZE];

	zCoeff[0] = -2.927222222 / (dz * dz);
  	zCoeff[1] = 1.666666667 / (dz * dz);
  	zCoeff[2] = -0.238095238 / (dz * dz);
  	zCoeff[3] = 0.039682539 / (dz * dz);
  	zCoeff[4] = -0.004960317 / (dz * dz);
  	zCoeff[5] = 0.000317460 / (dz * dz);

  	xCoeff[0] = -2.927222222 / (dx * dx);
  	xCoeff[1] = 1.666666667 / (dx * dx);
  	xCoeff[2] = -0.238095238 / (dx * dx);
  	xCoeff[3] = 0.039682539 / (dx * dx);
  	xCoeff[4] = -0.004960317 / (dx * dx);
  	xCoeff[5] = 0.000317460 / (dx * dx);

	/**************************** COMPUTE TIME-INTERPOLATION FILTER *********************/
	int hInterpFilter = sub + 1;
	int nInterpFilter = 2 * hInterpFilter;

	// Check the subsampling coefficient is smaller than the maximum allowed
	if (sub>=SUB_MAX){
		std::cout << "**** ERROR: Subsampling parameter too high ****" << std::endl;
		assert (1==2);
	}

	// Allocate and fill interpolation filter
	double interpFilter[nInterpFilter];
	for (int iFilter = 0; iFilter < hInterpFilter; iFilter++){
		interpFilter[iFilter] = 1.0 - 1.0 * iFilter/host_sub;
		interpFilter[iFilter+hInterpFilter] = 1.0 - interpFilter[iFilter];
		interpFilter[iFilter] = interpFilter[iFilter] * (1.0 / sqrt(double(host_ntw)/double(host_nts)));
		interpFilter[iFilter+hInterpFilter] = interpFilter[iFilter+hInterpFilter] * (1.0 / sqrt(double(host_ntw)/double(host_nts)));
	}

	/************************* COMPUTE COSINE DAMPING COEFFICIENTS **********************/
	if (minPad>=PAD_MAX){
		std::cout << "**** ERROR: Padding value is too high ****" << std::endl;
		assert (1==2);
	}
	double cosDampingCoeff[minPad];

	// Cosine padding
	for (int iFilter=FAT; iFilter<FAT+minPad; iFilter++){
		double arg = M_PI / (1.0 * minPad) * 1.0 * (minPad-iFilter+FAT);
		arg = alphaCos + (1.0-alphaCos) * cos(arg);
		cosDampingCoeff[iFilter-FAT] = arg;
	}

	// Check that the block size is consistent between parfile and "varDeclare.h"
	if (blockSize != BLOCK_SIZE) {
		std::cout << "**** ERROR: Block size for time stepper is not consistent with parfile ****" << std::endl;
		assert (1==2);
	}

	/**************************** COPY TO CONSTANT MEMORY *******************************/
	// Laplacian coefficients
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_zCoeff), zCoeff, COEFF_SIZE*sizeof(double), 0, hipMemcpyHostToDevice)); // Copy Laplacian coefficients to device
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_xCoeff), xCoeff, COEFF_SIZE*sizeof(double), 0, hipMemcpyHostToDevice));

	// Time interpolation filter
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nInterpFilter), &nInterpFilter, sizeof(int), 0, hipMemcpyHostToDevice)); // Filter length
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_hInterpFilter), &hInterpFilter, sizeof(int), 0, hipMemcpyHostToDevice)); // Filter half-length
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_interpFilter), interpFilter, nInterpFilter*sizeof(double), 0, hipMemcpyHostToDevice)); // Filter

	// Cosine damping parameters
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_cosDampingCoeff), &cosDampingCoeff, minPad*sizeof(double), 0, hipMemcpyHostToDevice)); // Array for damping
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_alphaCos), &alphaCos, sizeof(double), 0, hipMemcpyHostToDevice)); // Coefficient in the damping formula
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_minPad), &minPad, sizeof(int), 0, hipMemcpyHostToDevice)); // min (zPadMinus, zPadPlus, xPadMinus, xPadPlus)

	// FD parameters
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nz), &nz, sizeof(int), 0, hipMemcpyHostToDevice)); // Copy model size to device
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nx), &nx, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nts), &nts, sizeof(int), 0, hipMemcpyHostToDevice)); // Copy number of coarse time parameters to device
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_sub), &sub, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_ntw), &host_ntw, sizeof(int), 0, hipMemcpyHostToDevice)); // Copy number of coarse time parameters to device

	// Extension parameters
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nExt), &host_nExt, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_hExt), &host_hExt, sizeof(int), 0, hipMemcpyHostToDevice));

	// Second order time derivative coefficients
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_cCenter), &host_cCenter, sizeof(double), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_cSide), &host_cSide, sizeof(double), 0, hipMemcpyHostToDevice));

}

// Allocate on device
void allocateTomoExtShotsGpu(double *vel2Dtw2, double *reflectivityScale, double *extReflectivity, int iGpu){

	// Set GPU number
	hipSetDevice(iGpu);

	// Velocity scale
	cuda_call(hipMalloc((void**) &dev_vel2Dtw2[iGpu], host_nz*host_nx*sizeof(double))); // Allocate scaled velocity model on device
	cuda_call(hipMemcpy(dev_vel2Dtw2[iGpu], vel2Dtw2, host_nz*host_nx*sizeof(double), hipMemcpyHostToDevice));
	cuda_call(hipMemcpy(vel2Dtw2, dev_vel2Dtw2[iGpu], host_nz*host_nx*sizeof(double), hipMemcpyDeviceToHost));

	// Reflectivity scale
	cuda_call(hipMalloc((void**) &dev_reflectivityScale[iGpu], host_nz*host_nx*sizeof(double))); // Allocate scaling for reflectivity
	cuda_call(hipMemcpy(dev_reflectivityScale[iGpu], reflectivityScale, host_nz*host_nx*sizeof(double), hipMemcpyHostToDevice));

	// Allocate time slices
	cuda_call(hipMalloc((void**) &dev_p0[iGpu], host_nz*host_nx*sizeof(double))); // Fine time sampling temporary slices
	cuda_call(hipMalloc((void**) &dev_p1[iGpu], host_nz*host_nx*sizeof(double)));

  	cuda_call(hipMalloc((void**) &dev_ssLeft[iGpu], host_nz*host_nx*sizeof(double))); // Coarse time sampling temporary slices
  	cuda_call(hipMalloc((void**) &dev_ssRight[iGpu], host_nz*host_nx*sizeof(double)));

	cuda_call(hipMalloc((void**) &dev_scatLeft[iGpu], host_nz*host_nx*sizeof(double))); // Coarse time sampling temporary slices
  	cuda_call(hipMalloc((void**) &dev_scatRight[iGpu], host_nz*host_nx*sizeof(double)));

	cuda_call(hipMalloc((void**) &dev_ss0[iGpu], host_nz*host_nx*sizeof(double)));
	cuda_call(hipMalloc((void**) &dev_ss1[iGpu], host_nz*host_nx*sizeof(double)));
	cuda_call(hipMalloc((void**) &dev_ss2[iGpu], host_nz*host_nx*sizeof(double)));

	// Allocate non-extended model
	cuda_call(hipMalloc((void**) &dev_modelTomo[iGpu], host_nz*host_nx*sizeof(double)));

	// Allocate and copy from host to device extended reflectivity
	cuda_call(hipMalloc((void**) &dev_extReflectivity[iGpu], host_nz*host_nx*host_nExt*sizeof(double)));
	cuda_call(hipMemcpy(dev_extReflectivity[iGpu], extReflectivity, host_nz*host_nx*host_nExt*sizeof(double), hipMemcpyHostToDevice));

	// Scale reflectivity
	int nBlockZExt = (host_nz-2*FAT) / BLOCK_SIZE_EXT; // Number of blocks for the z-axis when using a time-extension
	int nBlockXExt = (host_nx-2*FAT) / BLOCK_SIZE_EXT; // Number of blocks for the x-axis when using a time-extension
	int nBlockExt = (host_nExt+BLOCK_SIZE_EXT-1) / BLOCK_SIZE_EXT;
	dim3 dimGridExt(nBlockZExt, nBlockXExt, nBlockExt);
	dim3 dimBlockExt(BLOCK_SIZE_EXT, BLOCK_SIZE_EXT, BLOCK_SIZE_EXT);
	kernel_exec(scaleReflectivityExt<<<dimGridExt, dimBlockExt>>>(dev_extReflectivity[iGpu], dev_reflectivityScale[iGpu], dev_vel2Dtw2[iGpu]));

	// Allocate source wavefield - The allocation of secondary sources are done inside the function implementation
	cuda_call(hipMalloc((void**) &dev_tomoSrcWavefieldDt2[iGpu], host_nz*host_nx*host_nts*sizeof(double))); // We store the source wavefield

	// Allocate the auxiliary wavefield (either scattered for the fwd, or receiver for the adjoint)
	cuda_call(hipMalloc((void**) &dev_tomoSecWavefield1[iGpu], host_nz*host_nx*host_nts*sizeof(double))); // We store the source wavefield
}

// Deallocate from device
void deallocateTomoExtShotsGpu(int iGpu){

 		// Set device number on GPU cluster
		hipSetDevice(iGpu);

		// Deallocate all DAT shit
    	cuda_call(hipFree(dev_vel2Dtw2[iGpu]));
    	cuda_call(hipFree(dev_reflectivityScale[iGpu]));
		cuda_call(hipFree(dev_p0[iGpu]));
    	cuda_call(hipFree(dev_p1[iGpu]));
		cuda_call(hipFree(dev_ssLeft[iGpu]));
		cuda_call(hipFree(dev_ssRight[iGpu]));
		cuda_call(hipFree(dev_ss0[iGpu]));
		cuda_call(hipFree(dev_ss1[iGpu]));
		cuda_call(hipFree(dev_ss2[iGpu]));
		cuda_call(hipFree(dev_tomoSrcWavefieldDt2[iGpu]));
		cuda_call(hipFree(dev_tomoSecWavefield1[iGpu]));
		cuda_call(hipFree(dev_extReflectivity[iGpu]));
		cuda_call(hipFree(dev_modelTomo[iGpu]));
}

/****************************************************************************************/
/************************************** Tomo forward ************************************/
/****************************************************************************************/
void tomoExtShotsFwdGpu(double *model, double *dataRegDts, double *sourcesSignals, int *sourcesPositionReg, int nSourcesReg, int *receiversPositionReg, int nReceiversReg, double *tomoSrcWavefieldDt2, double *tomoSecWavefield1, double *tomoSecWavefield2, int iGpu, int saveWavefield, std::string extension){

	// We assume the source wavelet/signals already contain(s) the second time derivative
	// Set device number
	hipSetDevice(iGpu);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(int)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(int), hipMemcpyHostToDevice));

	// Sources signals
  	cuda_call(hipMalloc((void**) &dev_sourcesSignals[iGpu], nSourcesReg*host_ntw*sizeof(double))); // Allocate sources signals on device
	cuda_call(hipMemcpy(dev_sourcesSignals[iGpu], sourcesSignals, nSourcesReg*host_ntw*sizeof(double), hipMemcpyHostToDevice)); // Copy sources signals on device

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(int)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(int), hipMemcpyHostToDevice));

	// Non-extended blocks/threads
	int nBlockZ = (host_nz-2*FAT) / BLOCK_SIZE; // Number of blocks for the z-axis
	int nBlockX = (host_nx-2*FAT) / BLOCK_SIZE; // Number of blocks for the x-axis
	int nBlockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA; // Number of blocks for the data extraction/injection
	dim3 dimGrid(nBlockZ, nBlockX);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	/************************************************************************************/
	/*************************************** Source *************************************/
	/************************************************************************************/
	// Compute source wavefield with second-order time derivative
	computeTomoSrcWfldDt2(dev_sourcesSignals[iGpu], dev_tomoSrcWavefieldDt2[iGpu], dev_sourcesPositionReg[iGpu], nSourcesReg, dimGrid, dimBlock, iGpu);

	// Copy source wavefield back to host
	if (saveWavefield == 1) {cuda_call(hipMemcpy(tomoSrcWavefieldDt2, dev_tomoSrcWavefieldDt2[iGpu], host_nz*host_nx*host_nts*sizeof(double), hipMemcpyDeviceToHost));}

	/************************************************************************************/
	/***************************** Preliminary steps ************************************/
	/************************************************************************************/
	// Copy + scale model (background perturbation)
	cuda_call(hipMemcpy(dev_modelTomo[iGpu], model, host_nz*host_nx*sizeof(double), hipMemcpyHostToDevice));
	kernel_exec(scaleReflectivity<<<dimGrid, dimBlock>>>(dev_modelTomo[iGpu], dev_reflectivityScale[iGpu], dev_vel2Dtw2[iGpu]));

	// Allocate data and initialize to zero
	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double)));
	cuda_call(hipMemset(dev_dataRegDts[iGpu], 0, nReceiversReg*host_nts*sizeof(double)));

	// Allocate secondary wavefields
	if (saveWavefield == 1) {cuda_call(hipMalloc((void**) &dev_tomoSecWavefield2[iGpu], host_nz*host_nx*host_nts*sizeof(double)));}

	/************************************************************************************/
	/************************************ Leg #1 ****************************************/
	/************************************************************************************/
	if (host_leg1 == 1){

		// source->reflectivity->model->data
		if (extension == "time") {

			computeTomoLeg1TimeFwd(dev_modelTomo[iGpu], dev_tomoSrcWavefieldDt2[iGpu], dev_dataRegDts[iGpu], dev_tomoSecWavefield1[iGpu], dev_tomoSecWavefield2[iGpu], dev_extReflectivity[iGpu], dev_receiversPositionReg[iGpu], nReceiversReg, dimGrid, dimBlock, nBlockData, iGpu, saveWavefield);

		} else {

			computeTomoLeg1OffsetFwd(dev_modelTomo[iGpu], dev_tomoSrcWavefieldDt2[iGpu], dev_dataRegDts[iGpu], dev_tomoSecWavefield1[iGpu], dev_tomoSecWavefield2[iGpu], dev_extReflectivity[iGpu], dev_receiversPositionReg[iGpu], nReceiversReg, dimGrid, dimBlock, nBlockData, iGpu, saveWavefield);

		}

		// Copy both scattered wavefields from leg #1 to host
		if (saveWavefield == 1) {
			cuda_call(hipMemcpy(tomoSecWavefield1, dev_tomoSecWavefield1[iGpu], host_nz*host_nx*host_nts*sizeof(double), hipMemcpyDeviceToHost));
			cuda_call(hipMemcpy(tomoSecWavefield2, dev_tomoSecWavefield2[iGpu], host_nz*host_nx*host_nts*sizeof(double), hipMemcpyDeviceToHost));
		}
	}

	/************************************************************************************/
	/************************************ Leg #2 ****************************************/
	/************************************************************************************/
	if (host_leg2 == 1){

		// source->model->reflectivity->data
		if (extension == "time"){

			computeTomoLeg2TimeFwd(dev_modelTomo[iGpu], dev_tomoSrcWavefieldDt2[iGpu], dev_dataRegDts[iGpu], dev_tomoSecWavefield1[iGpu], dev_tomoSecWavefield2[iGpu], dev_extReflectivity[iGpu], dev_receiversPositionReg[iGpu], nReceiversReg, dimGrid, dimBlock, nBlockData, iGpu, saveWavefield);

		} else {

			computeTomoLeg2OffsetFwd(dev_modelTomo[iGpu], dev_tomoSrcWavefieldDt2[iGpu], dev_dataRegDts[iGpu], dev_tomoSecWavefield1[iGpu], dev_tomoSecWavefield2[iGpu], dev_extReflectivity[iGpu], dev_receiversPositionReg[iGpu], nReceiversReg, dimGrid, dimBlock, nBlockData, iGpu, saveWavefield);

		}

		// Copy scattered wavefields from leg #2 to host
		if (saveWavefield == 1) {
			cuda_call(hipMemcpy(tomoSecWavefield1, dev_tomoSecWavefield1[iGpu], host_nz*host_nx*host_nts*sizeof(double), hipMemcpyDeviceToHost));
			cuda_call(hipMemcpy(tomoSecWavefield2, dev_tomoSecWavefield2[iGpu], host_nz*host_nx*host_nts*sizeof(double), hipMemcpyDeviceToHost));
		}
	}

	/************************************************************************************/
	/************************************ Data ******************************************/
	/************************************************************************************/
	// Copy data to host
	cuda_call(hipMemcpy(dataRegDts, dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double), hipMemcpyDeviceToHost));

	/******************************* Deallocation ***************************************/
	// Deallocate all slices
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_sourcesSignals[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));
    cuda_call(hipFree(dev_dataRegDts[iGpu]));
	if (saveWavefield == 1){ cuda_call(hipFree(dev_tomoSecWavefield2[iGpu]));}

}

/****************************************************************************************/
/************************************** Tomo adjoint ************************************/
/****************************************************************************************/
void tomoExtShotsAdjGpu(double *model, double *dataRegDts, double *sourcesSignals, int *sourcesPositionReg, int nSourcesReg, int *receiversPositionReg, int nReceiversReg, double *tomoSrcWavefieldDt2, double *tomoSecWavefield1, double *tomoSecWavefield2, int iGpu, int saveWavefield, std::string extension){

	// We assume the source wavelet/signals already contain(s) the second time derivative
	// Set device number
	hipSetDevice(iGpu);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(int)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(int), hipMemcpyHostToDevice));

	// Sources signals
  	cuda_call(hipMalloc((void**) &dev_sourcesSignals[iGpu], nSourcesReg*host_ntw*sizeof(double))); // Allocate sources signals on device
	cuda_call(hipMemcpy(dev_sourcesSignals[iGpu], sourcesSignals, nSourcesReg*host_ntw*sizeof(double), hipMemcpyHostToDevice)); // Copy sources signals on device

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(int)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(int), hipMemcpyHostToDevice));

	// Data
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double)));
	cuda_call(hipMemcpy(dev_dataRegDts[iGpu], dataRegDts, nReceiversReg*host_nts*sizeof(double), hipMemcpyHostToDevice));

	// Initialize models
	cuda_call(hipMemset(dev_modelTomo[iGpu], 0, host_nz*host_nx*sizeof(double)));

	// Non-extended blocks/threads
	int nBlockZ = (host_nz-2*FAT) / BLOCK_SIZE; // Number of blocks for the z-axis
	int nBlockX = (host_nx-2*FAT) / BLOCK_SIZE; // Number of blocks for the x-axis
	int nBlockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA; // Number of blocks for the data extraction/injection
	dim3 dimGrid(nBlockZ, nBlockX);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

   	// Extended blocks/threads
	int nBlockZExt = (host_nz-2*FAT) / BLOCK_SIZE_EXT; // Number of blocks for the z-axis when using a time-extension
	int nBlockXExt = (host_nx-2*FAT) / BLOCK_SIZE_EXT; // Number of blocks for the x-axis when using a time-extension
	int nBlockExt = (host_nExt+BLOCK_SIZE_EXT-1) / BLOCK_SIZE_EXT;
	dim3 dimGridExt(nBlockZExt, nBlockXExt, nBlockExt);
	dim3 dimBlockExt(BLOCK_SIZE_EXT, BLOCK_SIZE_EXT, BLOCK_SIZE_EXT);

	/************************************************************************************/
	/*************************************** Source *************************************/
	/************************************************************************************/
	// Compute source wavefield with second-order time derivative
	computeTomoSrcWfldDt2(dev_sourcesSignals[iGpu], dev_tomoSrcWavefieldDt2[iGpu], dev_sourcesPositionReg[iGpu], nSourcesReg, dimGrid, dimBlock, iGpu);
	if (saveWavefield == 1) {cuda_call(hipMemcpy(tomoSrcWavefieldDt2, dev_tomoSrcWavefieldDt2[iGpu], host_nz*host_nx*host_nts*sizeof(double), hipMemcpyDeviceToHost));}

	/************************************************************************************/
	/********************************** Receiver ****************************************/
	/************************************************************************************/
	// Compute receiver wavefield (include no time derivative)
	computeTomoRecWfld(dev_dataRegDts[iGpu], dev_tomoSecWavefield1[iGpu], dev_receiversPositionReg[iGpu], dimGrid, dimBlock, nBlockData, iGpu);
	if (saveWavefield == 1) {cuda_call(hipMemcpy(tomoSecWavefield1, dev_tomoSecWavefield1[iGpu], host_nz*host_nx*host_nts*sizeof(double), hipMemcpyDeviceToHost));}

	/************************************************************************************/
	/***************************** Preliminary steps ************************************/
	/************************************************************************************/
	// Allocate second secondary wavefield
	if (saveWavefield == 1) {cuda_call(hipMalloc((void**) &dev_tomoSecWavefield2[iGpu], host_nz*host_nx*host_nts*sizeof(double)));}

	// Initialize model for both legs
    cuda_call(hipMemset(dev_modelTomo[iGpu], 0, host_nz*host_nx*sizeof(double)));

	/************************************************************************************/
	/************************************ Leg #1 ****************************************/
	/************************************************************************************/
	if (host_leg1 == 1){

		if (extension == "time"){
			computeTomoLeg1TimeAdj(dev_tomoSrcWavefieldDt2[iGpu], dev_tomoSecWavefield1[iGpu], dev_modelTomo[iGpu], dev_tomoSecWavefield2[iGpu], dev_extReflectivity[iGpu], dimGrid, dimBlock, iGpu, saveWavefield);
		} else {
			computeTomoLeg1OffsetAdj(dev_tomoSrcWavefieldDt2[iGpu], dev_tomoSecWavefield1[iGpu], dev_modelTomo[iGpu], dev_tomoSecWavefield2[iGpu], dev_extReflectivity[iGpu], dimGrid, dimBlock, iGpu, saveWavefield);
		}

		// Copy scattered wavefield from leg #1 to host
		if (saveWavefield == 1) {cuda_call(hipMemcpy(tomoSecWavefield2, dev_tomoSecWavefield2[iGpu], host_nz*host_nx*host_nts*sizeof(double), hipMemcpyDeviceToHost));}

	}

	/************************************************************************************/
	/************************************ Leg #2 ****************************************/
	/************************************************************************************/
	if (host_leg2 == 1){

		if (extension == "time"){
			computeTomoLeg2TimeAdj(dev_tomoSrcWavefieldDt2[iGpu], dev_tomoSecWavefield1[iGpu], dev_modelTomo[iGpu], dev_tomoSecWavefield2[iGpu], dev_extReflectivity[iGpu], dimGrid, dimBlock, iGpu, saveWavefield);
		} else {
			computeTomoLeg2OffsetAdj(dev_tomoSrcWavefieldDt2[iGpu], dev_tomoSecWavefield1[iGpu], dev_modelTomo[iGpu], dev_tomoSecWavefield2[iGpu], dev_extReflectivity[iGpu], dimGrid, dimBlock, iGpu, saveWavefield);
		}

		// Copy scattered wavefield from leg #2 to host
		if (saveWavefield == 1) {cuda_call(hipMemcpy(tomoSecWavefield2, dev_tomoSecWavefield2[iGpu], host_nz*host_nx*host_nts*sizeof(double), hipMemcpyDeviceToHost));}

	}

	/************************************************************************************/
	/************************************ Model *****************************************/
	/************************************************************************************/
	// Scale model
	kernel_exec(scaleReflectivity<<<dimGrid, dimBlock>>>(dev_modelTomo[iGpu], dev_reflectivityScale[iGpu], dev_vel2Dtw2[iGpu]));

	// Copy data to host
	cuda_call(hipMemcpy(model, dev_modelTomo[iGpu], host_nz*host_nx*sizeof(double), hipMemcpyDeviceToHost));

	/******************************* Deallocation ***************************************/
	// Deallocate model, data and wavefields
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_sourcesSignals[iGpu]));
    cuda_call(hipFree(dev_dataRegDts[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));
	if (saveWavefield == 1) {cuda_call(hipFree(dev_tomoSecWavefield2[iGpu]));}
}

/****************************************************************************************/
/********************************** Auxiliary functions *********************************/
/****************************************************************************************/

/************************************* Common parts *************************************/

// Source wavefield
void computeTomoSrcWfldDt2(double *dev_sourcesIn, double *dev_wavefieldOut, int *dev_sourcesPositionsRegIn, int nSourcesRegIn, dim3 dimGridIn, dim3 dimBlockIn, int iGpu){

	// Initialize wavefield on device
	cuda_call(hipMemset(dev_wavefieldOut, 0, host_nz*host_nx*host_nts*sizeof(double)));

	// Initialize time-slices for time stepping
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_ssLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));
   	cuda_call(hipMemset(dev_ss0[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_ss1[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_ss2[iGpu], 0, host_nz*host_nx*sizeof(double)));

    // Compute coarse source wavefield sample for its=0
	int its = 0;
	for (int it2 = 1; it2 < host_sub+1; it2++){

		// Compute fine time-step index
		int itw = its * host_sub + it2;

		// Step forward
		kernel_exec(stepFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

		// Inject source
		kernel_exec(injectSource<<<1, nSourcesRegIn>>>(dev_sourcesIn, dev_p0[iGpu], itw-1, dev_sourcesPositionsRegIn));

		// Damp wavefields
		kernel_exec(dampCosineEdge<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu]));

		// Spread energy into dev_ss1 and dev_ss2
		kernel_exec(interpFineToCoarseSlice<<<dimGridIn, dimBlockIn>>>(dev_ssLeft[iGpu], dev_ssRight[iGpu], dev_p0[iGpu], it2));

		// Switch pointers
		dev_temp1[iGpu] = dev_p0[iGpu];
		dev_p0[iGpu] = dev_p1[iGpu];
		dev_p1[iGpu] = dev_temp1[iGpu];
		dev_temp1[iGpu] = NULL;

	}

	// Copy ss1 (its=0)
	cuda_call(hipMemcpy(dev_ss1[iGpu], dev_ssLeft[iGpu], host_nz*host_nx*sizeof(double), hipMemcpyDeviceToDevice));

	// Switch coarse grid pointers
	dev_ssTemp1[iGpu] = dev_ssLeft[iGpu];
	dev_ssLeft[iGpu] = dev_ssRight[iGpu];
	dev_ssRight[iGpu] = dev_ssTemp1[iGpu];
	cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));
	dev_ssTemp1[iGpu] = NULL;

	for (int its=1; its<host_nts-1; its++){

	    for (int it2=1; it2<host_sub+1; it2++){

	        // Compute fine time-step index
	        int itw = its * host_sub + it2;

	        // Step forward
	        kernel_exec(stepFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

	        // Inject source
	        kernel_exec(injectSource<<<1, nSourcesRegIn>>>(dev_sourcesIn, dev_p0[iGpu], itw-1, dev_sourcesPositionsRegIn));

	        // Damp wavefields
	        kernel_exec(dampCosineEdge<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu]));

	        // Spread energy into dev_ssLeft and dev_ssRight
	        kernel_exec(interpFineToCoarseSlice<<<dimGridIn, dimBlockIn>>>(dev_ssLeft[iGpu], dev_ssRight[iGpu], dev_p0[iGpu], it2));

	        // Switch pointers
	        dev_temp1[iGpu] = dev_p0[iGpu];
	        dev_p0[iGpu] = dev_p1[iGpu];
	        dev_p1[iGpu] = dev_temp1[iGpu];
	        dev_temp1[iGpu] = NULL;

	    }

		// Copy ss2 (value of source wavefield at its
	    cuda_call(hipMemcpy(dev_ss2[iGpu], dev_ssLeft[iGpu], host_nz*host_nx*sizeof(double), hipMemcpyDeviceToDevice));

	    // Compute second order time derivative of source wavefield at its-1
	    kernel_exec(srcWfldSecondTimeDerivative<<<dimGridIn, dimBlockIn>>>(dev_wavefieldOut, dev_ss0[iGpu], dev_ss1[iGpu], dev_ss2[iGpu], its-1));

	    // Switch coarse time sampling pointers
	    dev_ssTemp1[iGpu] = dev_ssLeft[iGpu];
	    dev_ssLeft[iGpu] = dev_ssRight[iGpu];
	    dev_ssRight[iGpu] = dev_ssTemp1[iGpu];
	    cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));
	    dev_ssTemp1[iGpu] = NULL;

	    // Switch pointers for time derivative
	    dev_ssTemp2[iGpu] = dev_ss0[iGpu];
	    dev_ss0[iGpu] = dev_ss1[iGpu];
	    dev_ss1[iGpu] = dev_ss2[iGpu];
	    dev_ss2[iGpu] = dev_ssTemp2[iGpu];
	    dev_ssTemp2[iGpu] = NULL;
	}

	// Copy ssLeft to ss2 which corresponds to wavefield value (before time derivative) at nts-1
	cuda_call(hipMemcpy(dev_ss2[iGpu], dev_ssLeft[iGpu], host_nz*host_nx*sizeof(double), hipMemcpyDeviceToDevice));

	// Compute second order time derivative at nts-2
	kernel_exec(srcWfldSecondTimeDerivative<<<dimGridIn, dimBlockIn>>>(dev_wavefieldOut, dev_ss0[iGpu], dev_ss1[iGpu], dev_ss2[iGpu], host_nts-2));

	// Compute second order time derivative at nts-1 (now ss2 is in the middle of the stencil)
	cuda_call(hipMemset(dev_ss0[iGpu], 0, host_nz*host_nx*sizeof(double)));
	kernel_exec(srcWfldSecondTimeDerivative<<<dimGridIn, dimBlockIn>>>(dev_wavefieldOut, dev_ss0[iGpu], dev_ss2[iGpu], dev_ss1[iGpu], host_nts-1));

}

// Receiver wavefield
void computeTomoRecWfld(double *dev_dataIn, double *dev_wavefieldOut, int *dev_receiversPositionsRegIn, dim3 dimGridIn, dim3 dimBlockIn, int nBlockDataIn, int iGpu){

	// Initialize wavefield on device
	cuda_call(hipMemset(dev_wavefieldOut, 0, host_nz*host_nx*host_nts*sizeof(double)));

	// Initialize time-slices for time stepping
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(double)));

	// Start propagation
	for (int its = host_nts-2; its > -1; its--){

		for (int it2 = host_sub-1; it2 > -1; it2--){

			// Step forward in time
			kernel_exec(stepAdjGpu<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject data
			kernel_exec(interpInjectData<<<nBlockDataIn, BLOCK_SIZE_DATA>>>(dev_dataIn, dev_p0[iGpu], its, it2, dev_receiversPositionsRegIn));

			// Damp wavefield
			kernel_exec(dampCosineEdge<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Interpolate and save wavefield on device (the wavefield is not scaled)
			kernel_exec(interpWavefield<<<dimGridIn, dimBlockIn>>>(dev_wavefieldOut, dev_p0[iGpu], its, it2));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;
		}
	}
}

/*************************************** Forward ****************************************/

// Leg 1 forward [time]: s -> i -> m -> d
void computeTomoLeg1TimeFwd(double *dev_modelIn, double *dev_tomoSrcWavefieldDt2In, double *dev_dataOut, double *dev_wavefield1Out, double *dev_wavefield2Out, double *dev_extReflectivityIn, int *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int nBlockDataIn, int iGpu, int saveWavefield){

	/**************** First part of leg #1 ****************/

	// Initialize scattered wavefield #1
    cuda_call(hipMemset(dev_wavefield1Out, 0, host_nz*host_nx*host_nts*sizeof(double))); // Note: We need to initialize the wavefield because the kernel interpWavefiel uses +=

    // Initialize slices
    cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(double)));
    cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(double)));
    cuda_call(hipMemset(dev_ssLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));
    cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));

	// Compute secondary source from extended scattering condition for first coarse time index (its = 0)
    int its = 0;
	int iExtMin, iExtMax;
    iExtMin = (its+1-host_nts)/2;
    iExtMin = std::max(iExtMin, -host_hExt) + host_hExt;
    iExtMax = its/2;
    iExtMax = std::min(iExtMax, host_hExt) + host_hExt + 1; // Add 1 for the strict inequality in the "for loop"
    kernel_exec(imagingTimeFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_extReflectivityIn, dev_ssLeft[iGpu], dev_tomoSrcWavefieldDt2In, its, iExtMin, iExtMax)); // Apply extended FWD imaging condition

    // Start propagating scattered wavefield
    for (int its = 0; its < host_nts-1; its++){

        // Compute secondary source for first coarse time index (its+1)
        iExtMin = (its+2-host_nts)/2;
        iExtMin = std::max(iExtMin, -host_hExt) + host_hExt; // Lower bound for extended index
        iExtMax = (its+1)/2;
        iExtMax = std::min(iExtMax, host_hExt) + host_hExt + 1; // Upper bound for extended index
        kernel_exec(imagingTimeFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_extReflectivityIn, dev_ssRight[iGpu], dev_tomoSrcWavefieldDt2In, its+1, iExtMin, iExtMax)); // Apply time-extended FWD imaging condition

        for (int it2 = 1; it2 < host_sub+1; it2++){

            // Step forward
            kernel_exec(stepFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));
            kernel_exec(injectSecondarySource<<<dimGridIn, dimBlockIn>>>(dev_ssLeft[iGpu], dev_ssRight[iGpu], dev_p0[iGpu], it2-1));
            kernel_exec(dampCosineEdge<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu]));
            kernel_exec(interpWavefield<<<dimGridIn, dimBlockIn>>>(dev_wavefield1Out, dev_p0[iGpu], its, it2));

            // Switch pointers
            dev_temp1[iGpu] = dev_p0[iGpu];
            dev_p0[iGpu] = dev_p1[iGpu];
            dev_p1[iGpu] = dev_temp1[iGpu];
            dev_temp1[iGpu] = NULL;

        }

        // Switch pointers for secondary source
        dev_ssTemp1[iGpu] = dev_ssLeft[iGpu];
        dev_ssLeft[iGpu] = dev_ssRight[iGpu];
        dev_ssRight[iGpu] = dev_ssTemp1[iGpu];
        dev_ssTemp1[iGpu] = NULL;

        // Reinitialize slice for right coarse sample. We have to do it for the extended imaging condition because we update dev_ssRight with += in the kernel
        // We don't have to do it for the non-extended imaging condition because there is no +=
        cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));

    }

	/**************** Second part of leg #1 ****************/
	if (saveWavefield == 1) {cuda_call(hipMemset(dev_wavefield2Out, 0, host_nz*host_nx*host_nts*sizeof(double)));} // Note: We need to initialize the wavefield because the kernel interpWavefiel uses +=

	// Initialize slices
    cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(double)));
    cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(double)));
    cuda_call(hipMemset(dev_ssLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));
    cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));

	// Compute secondary source for first coarse time index (its=0)
    kernel_exec(imagingFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_modelIn, dev_ssLeft[iGpu], 0, dev_wavefield1Out));

    // Start propagating scattered wavefield
    for (int its = 0; its < host_nts-1; its++){

        // Compute secondary source for first coarse time index (its+1)
        kernel_exec(imagingFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_modelIn, dev_ssRight[iGpu], its+1, dev_wavefield1Out));

        for (int it2 = 1; it2 < host_sub+1; it2++){

            // Update wavefield
            kernel_exec(stepFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));
            kernel_exec(injectSecondarySource<<<dimGridIn, dimBlockIn>>>(dev_ssLeft[iGpu], dev_ssRight[iGpu], dev_p0[iGpu], it2-1));
            kernel_exec(dampCosineEdge<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu]));
            if (saveWavefield == 1) {kernel_exec(interpWavefield<<<dimGridIn, dimBlockIn>>>(dev_wavefield2Out, dev_p0[iGpu], its, it2));}
            kernel_exec(recordInterpData<<<nBlockDataIn, BLOCK_SIZE_DATA>>>(dev_p0[iGpu], dev_dataOut, its, it2, dev_receiversPositionRegIn));

            // Switch pointers L1
            dev_temp1[iGpu] = dev_p0[iGpu];
            dev_p0[iGpu] = dev_p1[iGpu];
            dev_p1[iGpu] = dev_temp1[iGpu];
            dev_temp1[iGpu] = NULL;

        }

        // Switch pointers for secondary source
        dev_ssTemp1[iGpu] = dev_ssLeft[iGpu];
        dev_ssLeft[iGpu] = dev_ssRight[iGpu];
        dev_ssRight[iGpu] = dev_ssTemp1[iGpu];
        dev_ssTemp1[iGpu] = NULL;
    }
}

// Leg 2 forward [time]: s -> m -> i -> d
void computeTomoLeg2TimeFwd(double *dev_modelIn, double *dev_tomoSrcWavefieldDt2In, double *dev_dataOut, double *dev_wavefield1Out, double *dev_wavefield2Out, double *dev_extReflectivityIn, int *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int nBlockDataIn, int iGpu, int saveWavefield){

	// Initialize scattered wavefield #1
    cuda_call(hipMemset(dev_wavefield1Out, 0, host_nz*host_nx*host_nts*sizeof(double)));

    // Initialize slices
    cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(double)));
    cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(double)));
    cuda_call(hipMemset(dev_ssLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));
    cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));

	/************************** Scattered wavefield #1 ************************/
    // Compute secondary source for first coarse time index (its=0)
    kernel_exec(imagingFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_modelIn, dev_ssLeft[iGpu], 0, dev_tomoSrcWavefieldDt2In));

    // Start propagating scattered wavefield
    for (int its = 0; its < host_nts-1; its++){

        // Compute secondary source for first coarse time index (its+1)
        kernel_exec(imagingFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_modelIn, dev_ssRight[iGpu], its+1, dev_tomoSrcWavefieldDt2In));

        for (int it2 = 1; it2 < host_sub+1; it2++){

            // Update wavefield
            kernel_exec(stepFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));
            kernel_exec(injectSecondarySource<<<dimGridIn, dimBlockIn>>>(dev_ssLeft[iGpu], dev_ssRight[iGpu], dev_p0[iGpu], it2-1));
            kernel_exec(dampCosineEdge<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu]));
            kernel_exec(interpWavefield<<<dimGridIn, dimBlockIn>>>(dev_wavefield1Out, dev_p0[iGpu], its, it2));

            // Switch pointers L1
            dev_temp1[iGpu] = dev_p0[iGpu];
            dev_p0[iGpu] = dev_p1[iGpu];
            dev_p1[iGpu] = dev_temp1[iGpu];
            dev_temp1[iGpu] = NULL;

        }

        // Switch pointers for secondary source
        dev_ssTemp1[iGpu] = dev_ssLeft[iGpu];
        dev_ssLeft[iGpu] = dev_ssRight[iGpu];
        dev_ssRight[iGpu] = dev_ssTemp1[iGpu];
        dev_ssTemp1[iGpu] = NULL;
    }

	/************************** Scattered wavefield #2 ************************/
    // Initialize scattered wavefield
    if (saveWavefield == 1) {cuda_call(hipMemset(dev_wavefield2Out, 0, host_nz*host_nx*host_nts*sizeof(double)));}

    // Initialize slices
    cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(double)));
    cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(double)));
    cuda_call(hipMemset(dev_ssLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));
    cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));

    // Compute secondary source from extended scattering condition for first coarse time index (its=0)
    int its = 0;
	int iExtMin, iExtMax;
    iExtMin = (its+1-host_nts)/2;
    iExtMin = std::max(iExtMin, -host_hExt) + host_hExt; // Modified
    iExtMax = its/2;
    iExtMax = std::min(iExtMax, host_hExt) + host_hExt + 1; // Add 1 for the strict inequality in the "for loop"
    kernel_exec(imagingTimeFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_extReflectivityIn, dev_ssLeft[iGpu], dev_wavefield1Out, its, iExtMin, iExtMax)); // Apply extended FWD imaging condition

    // Start propagating scattered wavefield
    for (int its = 0; its < host_nts-1; its++){

        // Compute secondary source for first coarse time index (its+1)
        iExtMin = (its+2-host_nts)/2;
        iExtMin = std::max(iExtMin, -host_hExt) + host_hExt; // Lower bound for extended index
        iExtMax = (its+1)/2;
        iExtMax = std::min(iExtMax, host_hExt) + host_hExt + 1; // Upper bound for extended index
        kernel_exec(imagingTimeFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_extReflectivityIn, dev_ssRight[iGpu], dev_wavefield1Out, its+1, iExtMin, iExtMax)); // Apply time-extended FWD imaging condition

        for (int it2 = 1; it2 < host_sub+1; it2++){

            // Step forward
            kernel_exec(stepFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));
            kernel_exec(injectSecondarySource<<<dimGridIn, dimBlockIn>>>(dev_ssLeft[iGpu], dev_ssRight[iGpu], dev_p0[iGpu], it2-1));
            kernel_exec(dampCosineEdge<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu]));
            if (saveWavefield == 1) {kernel_exec(interpWavefield<<<dimGridIn, dimBlockIn>>>(dev_wavefield2Out, dev_p0[iGpu], its, it2));}
            kernel_exec(recordInterpData<<<nBlockDataIn, BLOCK_SIZE_DATA>>>(dev_p0[iGpu], dev_dataOut, its, it2, dev_receiversPositionRegIn));

            // Switch pointers
            dev_temp1[iGpu] = dev_p0[iGpu];
            dev_p0[iGpu] = dev_p1[iGpu];
            dev_p1[iGpu] = dev_temp1[iGpu];
            dev_temp1[iGpu] = NULL;

        }

        // Switch pointers for secondary source
        dev_ssTemp1[iGpu] = dev_ssLeft[iGpu];
        dev_ssLeft[iGpu] = dev_ssRight[iGpu];
        dev_ssRight[iGpu] = dev_ssTemp1[iGpu];
        dev_ssTemp1[iGpu] = NULL;
        cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double))); // Reinitialize slice for coarse
    }

}

// Leg 1 forward [offset]: s -> i -> m -> d
void computeTomoLeg1OffsetFwd(double *dev_modelIn, double *dev_tomoSrcWavefieldDt2In, double *dev_dataOut, double *dev_wavefield1Out, double *dev_wavefield2Out, double *dev_extReflectivityIn, int *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int nBlockDataIn, int iGpu, int saveWavefield){

	/**************** First part of leg #1 ****************/

	// Initialize scattered wavefield #1
    cuda_call(hipMemset(dev_wavefield1Out, 0, host_nz*host_nx*host_nts*sizeof(double))); // Note: We need to initialize the wavefield because the kernel interpWavefiel uses +=

    // Initialize slices
    cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(double)));
    cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(double)));
    cuda_call(hipMemset(dev_ssLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));
    cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));

	// Compute secondary source from extended scattering condition for first coarse time index (its = 0)
    int its = 0;
	kernel_exec(imagingOffsetFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_extReflectivityIn, dev_ssLeft[iGpu], dev_tomoSrcWavefieldDt2In, its)); // Apply extended FWD imaging condition

    // Start propagating scattered wavefield
    for (int its = 0; its < host_nts-1; its++){

        // Compute secondary source for first coarse time index (its+1)
        kernel_exec(imagingOffsetFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_extReflectivityIn, dev_ssRight[iGpu], dev_tomoSrcWavefieldDt2In, its+1)); // Apply time-extended FWD imaging condition

        for (int it2 = 1; it2 < host_sub+1; it2++){

            // Step forward
            kernel_exec(stepFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));
            kernel_exec(injectSecondarySource<<<dimGridIn, dimBlockIn>>>(dev_ssLeft[iGpu], dev_ssRight[iGpu], dev_p0[iGpu], it2-1));
            kernel_exec(dampCosineEdge<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu]));
            kernel_exec(interpWavefield<<<dimGridIn, dimBlockIn>>>(dev_wavefield1Out, dev_p0[iGpu], its, it2));

            // Switch pointers
            dev_temp1[iGpu] = dev_p0[iGpu];
            dev_p0[iGpu] = dev_p1[iGpu];
            dev_p1[iGpu] = dev_temp1[iGpu];
            dev_temp1[iGpu] = NULL;

        }

        // Switch pointers for secondary source
        dev_ssTemp1[iGpu] = dev_ssLeft[iGpu];
        dev_ssLeft[iGpu] = dev_ssRight[iGpu];
        dev_ssRight[iGpu] = dev_ssTemp1[iGpu];
        dev_ssTemp1[iGpu] = NULL;

        // Reinitialize slice for right coarse sample. We have to do it for the extended imaging condition because we update dev_ssRight with += in the kernel
        // We don't have to do it for the non-extended imaging condition because there is no +=
        cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));

    }

	/**************** Second part of leg #1 ****************/
	if (saveWavefield == 1) {cuda_call(hipMemset(dev_wavefield2Out, 0, host_nz*host_nx*host_nts*sizeof(double)));} // Note: We need to initialize the wavefield because the kernel interpWavefiel uses +=

	// Initialize slices
    cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(double)));
    cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(double)));
    cuda_call(hipMemset(dev_ssLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));
    cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));

	// Compute secondary source for first coarse time index (its=0)
    kernel_exec(imagingFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_modelIn, dev_ssLeft[iGpu], 0, dev_wavefield1Out));

    // Start propagating scattered wavefield
    for (int its = 0; its < host_nts-1; its++){

        // Compute secondary source for first coarse time index (its+1)
        kernel_exec(imagingFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_modelIn, dev_ssRight[iGpu], its+1, dev_wavefield1Out));

        for (int it2 = 1; it2 < host_sub+1; it2++){

            // Update wavefield
            kernel_exec(stepFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));
            kernel_exec(injectSecondarySource<<<dimGridIn, dimBlockIn>>>(dev_ssLeft[iGpu], dev_ssRight[iGpu], dev_p0[iGpu], it2-1));
            kernel_exec(dampCosineEdge<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu]));
            if (saveWavefield == 1) {kernel_exec(interpWavefield<<<dimGridIn, dimBlockIn>>>(dev_wavefield2Out, dev_p0[iGpu], its, it2));}
            kernel_exec(recordInterpData<<<nBlockDataIn, BLOCK_SIZE_DATA>>>(dev_p0[iGpu], dev_dataOut, its, it2, dev_receiversPositionRegIn));

            // Switch pointers L1
            dev_temp1[iGpu] = dev_p0[iGpu];
            dev_p0[iGpu] = dev_p1[iGpu];
            dev_p1[iGpu] = dev_temp1[iGpu];
            dev_temp1[iGpu] = NULL;

        }

        // Switch pointers for secondary source
        dev_ssTemp1[iGpu] = dev_ssLeft[iGpu];
        dev_ssLeft[iGpu] = dev_ssRight[iGpu];
        dev_ssRight[iGpu] = dev_ssTemp1[iGpu];
        dev_ssTemp1[iGpu] = NULL;
    }
}

// Leg 2 forward [offset]: s -> m -> i -> d
void computeTomoLeg2OffsetFwd(double *dev_modelIn, double *dev_tomoSrcWavefieldDt2In, double *dev_dataOut, double *dev_wavefield1Out, double *dev_wavefield2Out, double *dev_extReflectivityIn, int *dev_receiversPositionRegIn, int nReceiversRegIn, dim3 dimGridIn, dim3 dimBlockIn, int nBlockDataIn, int iGpu, int saveWavefield){

	// Initialize scattered wavefield #1
    cuda_call(hipMemset(dev_wavefield1Out, 0, host_nz*host_nx*host_nts*sizeof(double)));

    // Initialize slices
    cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(double)));
    cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(double)));
    cuda_call(hipMemset(dev_ssLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));
    cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));

	/************************** Scattered wavefield #1 ************************/
    // Compute secondary source for first coarse time index (its=0)
    kernel_exec(imagingFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_modelIn, dev_ssLeft[iGpu], 0, dev_tomoSrcWavefieldDt2In));

    // Start propagating scattered wavefield
    for (int its = 0; its < host_nts-1; its++){

        // Compute secondary source for first coarse time index (its+1)
        kernel_exec(imagingFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_modelIn, dev_ssRight[iGpu], its+1, dev_tomoSrcWavefieldDt2In));

        for (int it2 = 1; it2 < host_sub+1; it2++){

            // Update wavefield
            kernel_exec(stepFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));
            kernel_exec(injectSecondarySource<<<dimGridIn, dimBlockIn>>>(dev_ssLeft[iGpu], dev_ssRight[iGpu], dev_p0[iGpu], it2-1));
            kernel_exec(dampCosineEdge<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu]));
            kernel_exec(interpWavefield<<<dimGridIn, dimBlockIn>>>(dev_wavefield1Out, dev_p0[iGpu], its, it2));

            dev_temp1[iGpu] = dev_p0[iGpu];
            dev_p0[iGpu] = dev_p1[iGpu];
            dev_p1[iGpu] = dev_temp1[iGpu];
            dev_temp1[iGpu] = NULL;

        }

        // Switch pointers for secondary source
        dev_ssTemp1[iGpu] = dev_ssLeft[iGpu];
        dev_ssLeft[iGpu] = dev_ssRight[iGpu];
        dev_ssRight[iGpu] = dev_ssTemp1[iGpu];
        dev_ssTemp1[iGpu] = NULL;
    }

	/************************** Scattered wavefield #2 ************************/
    // Initialize scattered wavefield
    if (saveWavefield == 1) {cuda_call(hipMemset(dev_wavefield2Out, 0, host_nz*host_nx*host_nts*sizeof(double)));}

    // Initialize slices
    cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(double)));
    cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(double)));
    cuda_call(hipMemset(dev_ssLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));
    cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));

    // Compute secondary source from extended scattering condition for first coarse time index (its=0)
    int its = 0;
    kernel_exec(imagingOffsetFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_extReflectivityIn, dev_ssLeft[iGpu], dev_wavefield1Out, its)); // Apply extended FWD imaging condition

    // Start propagating scattered wavefield
    for (int its = 0; its < host_nts-1; its++){

        // Compute secondary source for first coarse time index (its+1)
        kernel_exec(imagingOffsetFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_extReflectivityIn, dev_ssRight[iGpu], dev_wavefield1Out, its+1)); // Apply time-extended FWD imaging condition

        for (int it2 = 1; it2 < host_sub+1; it2++){

            // Step forward
            kernel_exec(stepFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));
            kernel_exec(injectSecondarySource<<<dimGridIn, dimBlockIn>>>(dev_ssLeft[iGpu], dev_ssRight[iGpu], dev_p0[iGpu], it2-1));
            kernel_exec(dampCosineEdge<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu]));
            if (saveWavefield == 1) {kernel_exec(interpWavefield<<<dimGridIn, dimBlockIn>>>(dev_wavefield2Out, dev_p0[iGpu], its, it2));}
            kernel_exec(recordInterpData<<<nBlockDataIn, BLOCK_SIZE_DATA>>>(dev_p0[iGpu], dev_dataOut, its, it2, dev_receiversPositionRegIn));

            // Switch pointers
            dev_temp1[iGpu] = dev_p0[iGpu];
            dev_p0[iGpu] = dev_p1[iGpu];
            dev_p1[iGpu] = dev_temp1[iGpu];
            dev_temp1[iGpu] = NULL;

        }

        // Switch pointers for secondary source
        dev_ssTemp1[iGpu] = dev_ssLeft[iGpu];
        dev_ssLeft[iGpu] = dev_ssRight[iGpu];
        dev_ssRight[iGpu] = dev_ssTemp1[iGpu];
        dev_ssTemp1[iGpu] = NULL;
        cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double))); // Reinitialize slice for coarse
    }

}

// Leg 1 adjoint [time]: s -> i -> m <- d
void computeTomoLeg1TimeAdj(double *dev_tomoSrcWavefieldDt2In, double *dev_tomoRecWavefieldIn, double *dev_modelTomoOut, double *dev_wavefield1Out, double *dev_extReflectivityIn, dim3 dimGridIn, dim3 dimBlockIn, int iGpu, int saveWavefield){

	// Scatter source off of extended reflectivity and cross-correlate with receiver wavefield

	// Initialize scattered wavefield
	if (saveWavefield == 1){ cuda_call(hipMemset(dev_wavefield1Out, 0, host_nz*host_nx*host_nts*sizeof(double))); }

	// Initialize time slices on device
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_ssLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));
	cuda_call(hipMemset(dev_scatLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_scatRight[iGpu], 0, host_nz*host_nx*sizeof(double)));

	// Compute secondary source for first coarse time index (its = 0)
	int its = 0;
	int iExtMin, iExtMax;
	iExtMin = (its+1-host_nts)/2;
	iExtMin = std::max(iExtMin, -host_hExt) + host_hExt;
	iExtMax = its/2;
	iExtMax = std::min(iExtMax, host_hExt) + host_hExt + 1; // Add 1 for the strict inequality in the "for loop"
	kernel_exec(imagingTimeFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_extReflectivityIn, dev_ssLeft[iGpu], dev_tomoSrcWavefieldDt2In, its, iExtMin, iExtMax)); // Apply extended FWD imaging condition

	// Start propagating scattered wavefield
	for (int its = 0; its < host_nts-1; its++){

		// Compute secondary source for first coarse time index (its+1)
		iExtMin = (its+2-host_nts)/2;
		iExtMin = std::max(iExtMin, -host_hExt) + host_hExt; // Lower bound for extended index
		iExtMax = (its+1)/2;
		iExtMax = std::min(iExtMax, host_hExt) + host_hExt + 1; // Upper bound for extended index
		kernel_exec(imagingTimeFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_extReflectivityIn, dev_ssRight[iGpu], dev_tomoSrcWavefieldDt2In, its+1, iExtMin, iExtMax)); // Apply time-extended FWD imaging condition

		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Step forward
			kernel_exec(stepFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject secondary source sample itw-1
			kernel_exec(injectSecondarySource<<<dimGridIn, dimBlockIn>>>(dev_ssLeft[iGpu], dev_ssRight[iGpu], dev_p0[iGpu], it2-1));

			// Damp wavefields
			kernel_exec(dampCosineEdge<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Interpolate fine time slice to coarse time slice
			kernel_exec(interpFineToCoarseSlice<<<dimGridIn, dimBlockIn>>>(dev_scatLeft[iGpu], dev_scatRight[iGpu], dev_p0[iGpu], it2));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		// Apply imaging condition at its
		kernel_exec(imagingAdjGpu<<<dimGridIn, dimBlockIn>>>(dev_modelTomoOut, dev_scatLeft[iGpu], dev_tomoRecWavefieldIn, its));

		// Copy slice at its to scattered wavefield
		if (saveWavefield == 1) {cuda_call(hipMemcpy(dev_wavefield1Out+its*host_nz*host_nx, dev_scatLeft[iGpu], host_nz*host_nx*sizeof(double), hipMemcpyDeviceToDevice));}

		// Switch pointers for secondary source
		dev_ssTemp1[iGpu] = dev_ssLeft[iGpu];
		dev_ssLeft[iGpu] = dev_ssRight[iGpu];
		dev_ssRight[iGpu] = dev_ssTemp1[iGpu];
		dev_ssTemp1[iGpu] = NULL;
  		cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));

		// Switch pointers scattered wavefield
		dev_scatTemp1[iGpu] = dev_scatLeft[iGpu];
		dev_scatLeft[iGpu] = dev_scatRight[iGpu];
		dev_scatRight[iGpu] = dev_scatTemp1[iGpu];
		dev_scatTemp1[iGpu] = NULL;
  		cuda_call(hipMemset(dev_scatRight[iGpu], 0, host_nz*host_nx*sizeof(double)));

	}

	// Compute imaging condition at last sample its=nts-1
	kernel_exec(imagingAdjGpu<<<dimGridIn, dimBlockIn>>>(dev_modelTomoOut, dev_scatLeft[iGpu], dev_tomoRecWavefieldIn, host_nts-1));

	// Copy slice at nts-1 to scattered wavefield
	if (saveWavefield == 1) {cuda_call(hipMemcpy(dev_wavefield1Out+(host_nts-1)*host_nz*host_nx, dev_scatLeft[iGpu], host_nz*host_nx*sizeof(double), hipMemcpyDeviceToDevice));}

}

// Leg 2 adjoint [time]: s-> m <- i <- d
void computeTomoLeg2TimeAdj(double *dev_tomoSrcWavefieldDt2In, double *dev_tomoRecWavefieldIn, double *dev_modelTomoOut, double *dev_wavefield1Out, double *dev_extReflectivityIn, dim3 dimGridIn, dim3 dimBlockIn, int iGpu, int saveWavefield){

	// Note: No need to initialize wavefield1out because we are doing memcpy

	// Initialize time slices on device
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_ssLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));
	cuda_call(hipMemset(dev_scatLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_scatRight[iGpu], 0, host_nz*host_nx*sizeof(double)));

	// Compute secondary source for its=nts-1
	int its = host_nts-1;
	int iExtMin, iExtMax;
    iExtMin = (-its)/2;
    iExtMin = std::max(iExtMin, -host_hExt) + host_hExt;
    iExtMax = (host_nts-1-its)/2;
    iExtMax = std::min(iExtMax, host_hExt) + host_hExt + 1; // Add 1 for the strict inequality in the "for loop"
    kernel_exec(imagingTimeTomoAdjGpu<<<dimGridIn, dimBlockIn>>>(dev_tomoRecWavefieldIn, dev_ssRight[iGpu], dev_extReflectivityIn, its, iExtMin, iExtMax)); // Apply extended FWD imaging condition

	// Start propagating scattered wavefield
	for (int its = host_nts-2; its > -1; its--){

		// Compute secondary source for its
	    iExtMin = (-its)/2;
	    iExtMin = std::max(iExtMin, -host_hExt) + host_hExt;
	    iExtMax = (host_nts-1-its)/2;
	    iExtMax = std::min(iExtMax, host_hExt) + host_hExt + 1; // Add 1 for the strict inequality in the "for loop"
	    kernel_exec(imagingTimeTomoAdjGpu<<<dimGridIn, dimBlockIn>>>(dev_tomoRecWavefieldIn, dev_ssLeft[iGpu], dev_extReflectivityIn, its, iExtMin, iExtMax)); // Apply extended FWD imaging condition

		for (int it2 = host_sub-1; it2 > -1; it2--){

			// Step forward
			kernel_exec(stepAdjGpu<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject secondary source
			kernel_exec(injectSecondarySource<<<dimGridIn, dimBlockIn>>>(dev_ssLeft[iGpu], dev_ssRight[iGpu], dev_p0[iGpu], it2+1));

			// Damp wavefields
			kernel_exec(dampCosineEdge<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Interpolate fine time slice to coarse time slice
			kernel_exec(interpFineToCoarseSlice<<<dimGridIn, dimBlockIn>>>(dev_scatLeft[iGpu], dev_scatRight[iGpu], dev_p0[iGpu], it2));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		// Apply imaging condition at its+1
		kernel_exec(imagingAdjGpu<<<dimGridIn, dimBlockIn>>>(dev_modelTomoOut, dev_scatRight[iGpu], dev_tomoSrcWavefieldDt2In, its+1));

		// Copy slice at its+1 to adjoint scattered wavefield
		if (saveWavefield == 1) {cuda_call(hipMemcpy(dev_wavefield1Out+(its+1)*host_nz*host_nx, dev_scatRight[iGpu], host_nz*host_nx*sizeof(double), hipMemcpyDeviceToDevice));}

		// Switch pointers for secondary source
		dev_ssTemp1[iGpu] = dev_ssRight[iGpu];
		dev_ssRight[iGpu] = dev_ssLeft[iGpu];
		dev_ssLeft[iGpu] = dev_ssTemp1[iGpu];
		dev_ssTemp1[iGpu] = NULL;
  		cuda_call(hipMemset(dev_ssLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));

		// Switch pointers scattered wavefield
		dev_scatTemp1[iGpu] = dev_scatRight[iGpu];
		dev_scatRight[iGpu] = dev_scatLeft[iGpu];
		dev_scatLeft[iGpu] = dev_scatTemp1[iGpu];
		dev_scatTemp1[iGpu] = NULL;
  		cuda_call(hipMemset(dev_scatLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));

	}

	// Compute imaging condition at last sample its=nts-1
	kernel_exec(imagingAdjGpu<<<dimGridIn, dimBlockIn>>>(dev_modelTomoOut, dev_scatRight[iGpu], dev_tomoSrcWavefieldDt2In, 0));

	// Copy slice at its=0 to adjoint scattered wavefield
	if (saveWavefield == 1) {cuda_call(hipMemcpy(dev_wavefield1Out, dev_scatRight[iGpu], host_nz*host_nx*sizeof(double), hipMemcpyDeviceToDevice));}

}

// Leg 1 adjoint [offset]: s -> i -> m <- d
void computeTomoLeg1OffsetAdj(double *dev_tomoSrcWavefieldDt2In, double *dev_tomoRecWavefieldIn, double *dev_modelTomoOut, double *dev_wavefield1Out, double *dev_extReflectivityIn, dim3 dimGridIn, dim3 dimBlockIn, int iGpu, int saveWavefield){

	// Initialize time slices on device
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_ssLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));
	cuda_call(hipMemset(dev_scatLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_scatRight[iGpu], 0, host_nz*host_nx*sizeof(double)));

	// Compute secondary source for first coarse time index (its = 0)
	int its = 0;
	kernel_exec(imagingOffsetFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_extReflectivityIn, dev_ssLeft[iGpu], dev_tomoSrcWavefieldDt2In, its)); // Apply extended FWD imaging condition

	// Start propagating scattered wavefield
	for (int its = 0; its < host_nts-1; its++){

		// Compute secondary source for first coarse time index (its+1)
		kernel_exec(imagingOffsetFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_extReflectivityIn, dev_ssRight[iGpu], dev_tomoSrcWavefieldDt2In, its+1)); // Apply time-extended FWD imaging condition

		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Step forward
			kernel_exec(stepFwdGpu<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject secondary source sample itw-1
			kernel_exec(injectSecondarySource<<<dimGridIn, dimBlockIn>>>(dev_ssLeft[iGpu], dev_ssRight[iGpu], dev_p0[iGpu], it2-1));

			// Damp wavefields
			kernel_exec(dampCosineEdge<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Interpolate fine time slice to coarse time slice
			kernel_exec(interpFineToCoarseSlice<<<dimGridIn, dimBlockIn>>>(dev_scatLeft[iGpu], dev_scatRight[iGpu], dev_p0[iGpu], it2));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		// Apply imaging condition at its
		kernel_exec(imagingAdjGpu<<<dimGridIn, dimBlockIn>>>(dev_modelTomoOut, dev_scatLeft[iGpu], dev_tomoRecWavefieldIn, its));

		// Copy slice at its to scattered wavefield
		if (saveWavefield == 1) {cuda_call(hipMemcpy(dev_wavefield1Out+its*host_nz*host_nx, dev_scatLeft[iGpu], host_nz*host_nx*sizeof(double), hipMemcpyDeviceToDevice));}

		// Switch pointers for secondary source
		dev_ssTemp1[iGpu] = dev_ssLeft[iGpu];
		dev_ssLeft[iGpu] = dev_ssRight[iGpu];
		dev_ssRight[iGpu] = dev_ssTemp1[iGpu];
		dev_ssTemp1[iGpu] = NULL;
  		cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));

		// Switch pointers scattered wavefield
		dev_scatTemp1[iGpu] = dev_scatLeft[iGpu];
		dev_scatLeft[iGpu] = dev_scatRight[iGpu];
		dev_scatRight[iGpu] = dev_scatTemp1[iGpu];
		dev_scatTemp1[iGpu] = NULL;
  		cuda_call(hipMemset(dev_scatRight[iGpu], 0, host_nz*host_nx*sizeof(double)));

	}

	// Compute imaging condition at last sample its=nts-1
	kernel_exec(imagingAdjGpu<<<dimGridIn, dimBlockIn>>>(dev_modelTomoOut, dev_scatLeft[iGpu], dev_tomoRecWavefieldIn, host_nts-1));

	// Copy slice at nts-1 to scattered wavefield
	if (saveWavefield == 1) {cuda_call(hipMemcpy(dev_wavefield1Out+(host_nts-1)*host_nz*host_nx, dev_scatLeft[iGpu], host_nz*host_nx*sizeof(double), hipMemcpyDeviceToDevice));}

}

// Leg 2 adjoint [offset]: s-> m <- i <- d
void computeTomoLeg2OffsetAdj(double *dev_tomoSrcWavefieldDt2In, double *dev_tomoRecWavefieldIn, double *dev_modelTomoOut, double *dev_wavefield1Out, double *dev_extReflectivityIn, dim3 dimGridIn, dim3 dimBlockIn, int iGpu, int saveWavefield){

	// Initialize time slices on device
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_ssLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));
	cuda_call(hipMemset(dev_scatLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_scatRight[iGpu], 0, host_nz*host_nx*sizeof(double)));

	// Compute secondary source for its=nts-1
	int its = host_nts-1;
    kernel_exec(imagingOffsetTomoAdjGpu<<<dimGridIn, dimBlockIn>>>(dev_tomoRecWavefieldIn, dev_ssRight[iGpu], dev_extReflectivityIn, its));

	// Start propagating scattered wavefield
	for (int its = host_nts-2; its > -1; its--){

		// Compute secondary source for its
	    kernel_exec(imagingOffsetTomoAdjGpu<<<dimGridIn, dimBlockIn>>>(dev_tomoRecWavefieldIn, dev_ssLeft[iGpu], dev_extReflectivityIn, its));

		for (int it2 = host_sub-1; it2 > -1; it2--){

			// Step forward
			kernel_exec(stepAdjGpu<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject secondary source
			kernel_exec(injectSecondarySource<<<dimGridIn, dimBlockIn>>>(dev_ssLeft[iGpu], dev_ssRight[iGpu], dev_p0[iGpu], it2+1));

			// Damp wavefields
			kernel_exec(dampCosineEdge<<<dimGridIn, dimBlockIn>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Interpolate fine time slice to coarse time slice
			kernel_exec(interpFineToCoarseSlice<<<dimGridIn, dimBlockIn>>>(dev_scatLeft[iGpu], dev_scatRight[iGpu], dev_p0[iGpu], it2));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		// Apply imaging condition at its+1
		kernel_exec(imagingAdjGpu<<<dimGridIn, dimBlockIn>>>(dev_modelTomoOut, dev_scatRight[iGpu], dev_tomoSrcWavefieldDt2In, its+1));

		// Copy slice at its+1 to adjoint scattered wavefield
		if (saveWavefield == 1) {cuda_call(hipMemcpy(dev_wavefield1Out+(its+1)*host_nz*host_nx, dev_scatRight[iGpu], host_nz*host_nx*sizeof(double), hipMemcpyDeviceToDevice));}

		// Switch pointers for secondary source
		dev_ssTemp1[iGpu] = dev_ssRight[iGpu];
		dev_ssRight[iGpu] = dev_ssLeft[iGpu];
		dev_ssLeft[iGpu] = dev_ssTemp1[iGpu];
		dev_ssTemp1[iGpu] = NULL;
  		cuda_call(hipMemset(dev_ssLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));

		// Switch pointers scattered wavefield
		dev_scatTemp1[iGpu] = dev_scatRight[iGpu];
		dev_scatRight[iGpu] = dev_scatLeft[iGpu];
		dev_scatLeft[iGpu] = dev_scatTemp1[iGpu];
		dev_scatTemp1[iGpu] = NULL;
  		cuda_call(hipMemset(dev_scatLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));

	}

	// Compute imaging condition at last sample its=nts-1
	kernel_exec(imagingAdjGpu<<<dimGridIn, dimBlockIn>>>(dev_modelTomoOut, dev_scatRight[iGpu], dev_tomoSrcWavefieldDt2In, 0));

	// Copy slice at its=0 to adjoint scattered wavefield
	if (saveWavefield == 1) {cuda_call(hipMemcpy(dev_wavefield1Out, dev_scatRight[iGpu], host_nz*host_nx*sizeof(double), hipMemcpyDeviceToDevice));}

}
