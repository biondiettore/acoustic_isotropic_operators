#include "hip/hip_runtime.h"
#include "BornShotsGpuFunctions.h"
#include <iostream>
#include "varDeclare.h"
#include <vector>
#include <algorithm>
#include <math.h>
#include "kernelsGpu.cu"
#include "cudaErrors.cu"
#include <stdio.h>
#include <assert.h>

/****************************************************************************************/
/******************************* Set GPU propagation parameters *************************/
/****************************************************************************************/
bool getGpuInfo(std::vector<int> gpuList, int info, int deviceNumberInfo){

	int nDevice, driver;
	hipGetDeviceCount(&nDevice);

	if (info == 1){

		std::cout << " " << std::endl;
		std::cout << "-------------------------------------------------------------------" << std::endl;
		std::cout << "---------------------------- INFO FOR GPU# " << deviceNumberInfo << " ----------------------" << std::endl;
		std::cout << "-------------------------------------------------------------------" << std::endl;

		// Number of devices
		std::cout << "Number of requested GPUs: " << gpuList.size() << std::endl;
		std::cout << "Number of available GPUs: " << nDevice << std::endl;
		std::cout << "Id of requested GPUs: ";
		for (int iGpu=0; iGpu<gpuList.size(); iGpu++){
			if (iGpu<gpuList.size()-1){std::cout << gpuList[iGpu] << ", ";}
 			else{ std::cout << gpuList[iGpu] << std::endl;}
		}

		// Driver version
		std::cout << "Cuda driver version: " << hipDriverGetVersion(&driver) << std::endl; // Driver

		// Get properties
		hipDeviceProp_t dprop;
		hipGetDeviceProperties(&dprop,deviceNumberInfo);

		// Display
		std::cout << "Name: " << dprop.name << std::endl;
		std::cout << "Total global memory: " << dprop.totalGlobalMem/(1024*1024*1024) << " [GB] " << std::endl;
		std::cout << "Shared memory per block: " << dprop.sharedMemPerBlock/1024 << " [kB]" << std::endl;
		std::cout << "Number of register per block: " << dprop.regsPerBlock << std::endl;
		std::cout << "Warp size: " << dprop.warpSize << " [threads]" << std::endl;
		std::cout << "Maximum pitch allowed for memory copies in bytes: " << dprop.memPitch/(1024*1024*1024) << " [GB]" << std::endl;
		std::cout << "Maximum threads per block: " << dprop.maxThreadsPerBlock << std::endl;
		std::cout << "Maximum block dimensions: " << "(" << dprop.maxThreadsDim[0] << ", " << dprop.maxThreadsDim[1] << ", " << dprop.maxThreadsDim[2] << ")" << std::endl;
		std::cout << "Maximum grid dimensions: " << "(" << dprop.maxGridSize[0] << ", " << dprop.maxGridSize[1] << ", " << dprop.maxGridSize[2] << ")" << std::endl;
		std::cout << "Total constant memory: " << dprop.totalConstMem/1024 << " [kB]" << std::endl;
		std::cout << "Number of streaming multiprocessors on device: " << dprop.multiProcessorCount << std::endl;
		if (dprop.deviceOverlap == 1) {std::cout << "Device can simultaneously perform a hipMemcpy() and kernel execution" << std::endl;}
		if (dprop.deviceOverlap != 1) {std::cout << "Device cannot simultaneously perform a hipMemcpy() and kernel execution" << std::endl;}
		if (dprop.canMapHostMemory == 1) { std::cout << "Device can map host memory" << std::endl; }
		if (dprop.canMapHostMemory != 1) { std::cout << "Device cannot map host memory" << std::endl; }
		if (dprop.concurrentKernels == 1) {std::cout << "Device can support concurrent kernel" << std::endl;}
		if (dprop.concurrentKernels != 1) {std::cout << "Device cannot support concurrent kernel execution" << std::endl;}

		std::cout << "-------------------------------------------------------------------" << std::endl;
		std::cout << " " << std::endl;
	}

	// Check that the number of requested GPU is less or equal to the total number of available GPUs
	if (gpuList.size()>nDevice) {
		std::cout << "**** ERROR [getGpuInfo]: Number of requested GPU greater than available GPUs ****" << std::endl;
		return false;
	}

	// Check that the GPU numbers in the list are between 0 and nGpu-1
	for (int iGpu=0; iGpu<gpuList.size(); iGpu++){
		if (gpuList[iGpu]<0 || gpuList[iGpu]>nDevice-1){
			std::cout << "**** ERROR [getGpuInfo]: One of the element of the GPU Id list is not a valid GPU Id number ****" << std::endl;
			return false;
		}
	}
	return true;
}
void initBornGpu(double dz, double dx, int nz, int nx, int nts, double dts, int sub, int minPad, int blockSize, double alphaCos, int nGpu, int iGpuId, int iGpuAlloc){

	// Set GPU number
	hipSetDevice(iGpuId);

	host_nz = nz;
	host_nx = nx;
	host_dz = dz;
	host_dx = dx;
	host_nts = nts;
	host_sub = sub;
	host_ntw = (nts - 1) * sub + 1;

	/**************************** ALLOCATE ARRAYS OF ARRAYS *****************************/
	// Only one GPU will perform the following
	if (iGpuId == iGpuAlloc) {

		// Time slices for FD stepping
		dev_p0 = new double*[nGpu];
		dev_p1 = new double*[nGpu];
		dev_temp1 = new double*[nGpu];

		dev_ssLeft = new double*[nGpu];
		dev_ssRight = new double*[nGpu];
		dev_ssTemp1 = new double*[nGpu];

		// Data
		dev_dataRegDts = new double*[nGpu];

		// Source and receivers
		dev_sourcesPositionReg = new int*[nGpu];
		dev_receiversPositionReg = new int*[nGpu];

		// Sources signal
		dev_sourcesSignals = new double*[nGpu];

		// Scaled velocity
		dev_vel2Dtw2 = new double*[nGpu];

		// Reflectivity scaling
		dev_reflectivityScale = new double*[nGpu];

		// Reflectivity
		dev_modelBorn = new double*[nGpu];

		// Source wavefields
		dev_BornSrcWavefield = new double*[nGpu];

	}

	/**************************** COMPUTE LAPLACIAN COEFFICIENTS ************************/
	double zCoeff[COEFF_SIZE];
	double xCoeff[COEFF_SIZE];

	zCoeff[0] = -2.927222222 / (dz * dz);
  	zCoeff[1] = 1.666666667 / (dz * dz);
  	zCoeff[2] = -0.238095238 / (dz * dz);
  	zCoeff[3] = 0.039682539 / (dz * dz);
  	zCoeff[4] = -0.004960317 / (dz * dz);
  	zCoeff[5] = 0.000317460 / (dz * dz);

  	xCoeff[0] = -2.927222222 / (dx * dx);
  	xCoeff[1] = 1.666666667 / (dx * dx);
  	xCoeff[2] = -0.238095238 / (dx * dx);
  	xCoeff[3] = 0.039682539 / (dx * dx);
  	xCoeff[4] = -0.004960317 / (dx * dx);
  	xCoeff[5] = 0.000317460 / (dx * dx);

	/**************************** COMPUTE TIME-INTERPOLATION FILTER *********************/
	// Time interpolation filter length/half length
	int hInterpFilter = sub + 1;
	int nInterpFilter = 2 * hInterpFilter;

	// Check the subsampling coefficient is smaller than the maximum allowed
	if (sub>SUB_MAX){
		std::cout << "**** ERROR: Subsampling parameter is too high ****" << std::endl;
		assert (1==2);
	}

	// Allocate and fill interpolation filter
	double interpFilter[nInterpFilter];
	for (int iFilter = 0; iFilter < hInterpFilter; iFilter++){
		interpFilter[iFilter] = 1.0 - 1.0 * iFilter/host_sub;
		interpFilter[iFilter+hInterpFilter] = 1.0 - interpFilter[iFilter];
		interpFilter[iFilter] = interpFilter[iFilter] * (1.0 / sqrt(double(host_ntw)/double(host_nts)));
		interpFilter[iFilter+hInterpFilter] = interpFilter[iFilter+hInterpFilter] * (1.0 / sqrt(double(host_ntw)/double(host_nts)));
	}

	/************************* COMPUTE COSINE DAMPING COEFFICIENTS **********************/
	// Check that the minimum padding is smaller than the max allowed
	if (minPad>PAD_MAX){
		std::cout << "**** ERROR: Padding value is too high ****" << std::endl;
		assert (1==2);
	}
	double cosDampingCoeff[minPad];

	// Cosine padding
	for (int iFilter=FAT; iFilter<FAT+minPad; iFilter++){
		double arg = M_PI / (1.0 * minPad) * 1.0 * (minPad-iFilter+FAT);
		arg = alphaCos + (1.0-alphaCos) * cos(arg);
		cosDampingCoeff[iFilter-FAT] = arg;
	}

	// Check that the block size is consistent between parfile and "varDeclare.h"
	if (blockSize != BLOCK_SIZE) {
		std::cout << "**** ERROR: Block size for time stepper is not consistent with parfile ****" << std::endl;
		assert (1==2);
	}

	/**************************** COPY TO CONSTANT MEMORY *******************************/
	// Laplacian coefficients
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_zCoeff), zCoeff, COEFF_SIZE*sizeof(double), 0, hipMemcpyHostToDevice)); // Copy Laplacian coefficients to device
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_xCoeff), xCoeff, COEFF_SIZE*sizeof(double), 0, hipMemcpyHostToDevice));

	// Time interpolation filter
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nInterpFilter), &nInterpFilter, sizeof(int), 0, hipMemcpyHostToDevice)); // Filter length
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_hInterpFilter), &hInterpFilter, sizeof(int), 0, hipMemcpyHostToDevice)); // Filter half-length
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_interpFilter), interpFilter, nInterpFilter*sizeof(double), 0, hipMemcpyHostToDevice)); // Filter

	// Cosine damping parameters
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_cosDampingCoeff), &cosDampingCoeff, minPad*sizeof(double), 0, hipMemcpyHostToDevice)); // Array for damping
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_alphaCos), &alphaCos, sizeof(double), 0, hipMemcpyHostToDevice)); // Coefficient in the damping formula
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_minPad), &minPad, sizeof(int), 0, hipMemcpyHostToDevice)); // min (zPadMinus, zPadPlus, xPadMinus, xPadPlus)

	// FD parameters
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nz), &nz, sizeof(int), 0, hipMemcpyHostToDevice)); // Copy model size to device
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nx), &nx, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nts), &nts, sizeof(int), 0, hipMemcpyHostToDevice)); // Copy number of coarse time parameters to device
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_sub), &sub, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_ntw), &host_ntw, sizeof(int), 0, hipMemcpyHostToDevice)); // Copy number of coarse time parameters to device

}
void allocateBornShotsGpu(double *vel2Dtw2, double *reflectivityScale, int iGpu, int iGpuId){

	// Set GPU number
	hipSetDevice(iGpuId);

	// Reflectivity scale
	cuda_call(hipMalloc((void**) &dev_vel2Dtw2[iGpu], host_nz*host_nx*sizeof(double))); // Allocate scaled velocity model on device
	cuda_call(hipMemcpy(dev_vel2Dtw2[iGpu], vel2Dtw2, host_nz*host_nx*sizeof(double), hipMemcpyHostToDevice)); //

	// Scaled velocity
	cuda_call(hipMalloc((void**) &dev_reflectivityScale[iGpu], host_nz*host_nx*sizeof(double))); // Allocate scaling for reflectivity
	cuda_call(hipMemcpy(dev_reflectivityScale[iGpu], reflectivityScale, host_nz*host_nx*sizeof(double), hipMemcpyHostToDevice)); //

	// Allocate time slices
	cuda_call(hipMalloc((void**) &dev_p0[iGpu], host_nz*host_nx*sizeof(double)));
	cuda_call(hipMalloc((void**) &dev_p1[iGpu], host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMalloc((void**) &dev_ssLeft[iGpu], host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMalloc((void**) &dev_ssRight[iGpu], host_nz*host_nx*sizeof(double)));

	// Allocate non-extended model
	cuda_call(hipMalloc((void**) &dev_modelBorn[iGpu], host_nz*host_nx*sizeof(double)));

	// Allocate source wavefield
	cuda_call(hipMalloc((void**) &dev_BornSrcWavefield[iGpu], host_nz*host_nx*host_nts*sizeof(double))); // Allocate on device

}
void deallocateBornShotsGpu(int iGpu, int iGpuId){

 		// Set device number on GPU cluster
		hipSetDevice(iGpuId);

		// Deallocate all the shit
    	cuda_call(hipFree(dev_vel2Dtw2[iGpu]));
    	cuda_call(hipFree(dev_reflectivityScale[iGpu]));
		cuda_call(hipFree(dev_p0[iGpu]));
    	cuda_call(hipFree(dev_p1[iGpu]));
		cuda_call(hipFree(dev_ssLeft[iGpu]));
		cuda_call(hipFree(dev_ssRight[iGpu]));
		cuda_call(hipFree(dev_BornSrcWavefield[iGpu]));
		cuda_call(hipFree(dev_modelBorn[iGpu]));
}

/****************************************************************************************/
/************************************** Born forward ************************************/
/****************************************************************************************/
void BornShotsFwdGpu(double *model, double *dataRegDts, double *sourcesSignals, int *sourcesPositionReg, int nSourcesReg, int *receiversPositionReg, int nReceiversReg, double *srcWavefieldDts, double *scatWavefieldDts, int iGpu, int iGpuId){

	// We assume the source wavelet/signals already contain the second time derivative
	// Set device number
	hipSetDevice(iGpuId);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(int)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(int), hipMemcpyHostToDevice));

	// Sources signals
  	cuda_call(hipMalloc((void**) &dev_sourcesSignals[iGpu], nSourcesReg*host_ntw*sizeof(double))); // Allocate sources signals on device
	cuda_call(hipMemcpy(dev_sourcesSignals[iGpu], sourcesSignals, nSourcesReg*host_ntw*sizeof(double), hipMemcpyHostToDevice)); // Copy sources signals on device

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(int)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(int), hipMemcpyHostToDevice));

	// Initialize source wavefield on device
	cuda_call(hipMemset(dev_BornSrcWavefield[iGpu], 0, host_nz*host_nx*host_nts*sizeof(double))); // Initialize wavefield on device

	// Initialize time-slices for time-stepping
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(double)));

   	// Kernel parameters
	int nblockx = (host_nz-2*FAT) / BLOCK_SIZE;
	int nblocky = (host_nx-2*FAT) / BLOCK_SIZE;
	int nblockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	/************************** Source wavefield computation ****************************/

	for (int its = 0; its < host_nts-1; its++){
		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Compute fine time-step index
			int itw = its * host_sub + it2;

			// Step forward
			kernel_exec(stepFwdGpu<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject source
			kernel_exec(injectSource<<<1, nSourcesReg>>>(dev_sourcesSignals[iGpu], dev_p0[iGpu], itw-1, dev_sourcesPositionReg[iGpu]));

			// Damp wavefields
			kernel_exec(dampCosineEdge<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Extract wavefield
			kernel_exec(interpWavefield<<<dimGrid, dimBlock>>>(dev_BornSrcWavefield[iGpu], dev_p0[iGpu], its, it2));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}
	}

	/************************** Scattered wavefield computation *************************/
	// Initialize time slices on device
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_ssLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));

	// Copy model to device
	cuda_call(hipMemcpy(dev_modelBorn[iGpu], model, host_nz*host_nx*sizeof(double), hipMemcpyHostToDevice)); // Copy model (reflectivity) on device

	// Allocate and initialize data
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double))); // Allocate data at coarse time-sampling on device
  	cuda_call(hipMemset(dev_dataRegDts[iGpu], 0, nReceiversReg*host_nts*sizeof(double))); // Initialize data on device

	// Apply both scalings to reflectivity: (1) 2.0*1/v^3 (2) v^2*dtw^2
	kernel_exec(scaleReflectivity<<<dimGrid, dimBlock>>>(dev_modelBorn[iGpu], dev_reflectivityScale[iGpu], dev_vel2Dtw2[iGpu]));

	// Compute secondary source for first coarse time index (its = 0)
	kernel_exec(imagingFwdGpu<<<dimGrid, dimBlock>>>(dev_modelBorn[iGpu], dev_ssLeft[iGpu], 0, dev_BornSrcWavefield[iGpu]));

	// Start propagating scattered wavefield
	for (int its = 0; its < host_nts-1; its++){

		// Compute secondary source for first coarse time index (its+1)
		kernel_exec(imagingFwdGpu<<<dimGrid, dimBlock>>>(dev_modelBorn[iGpu], dev_ssRight[iGpu], its+1, dev_BornSrcWavefield[iGpu]));

		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Step forward
			kernel_exec(stepFwdGpu<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject secondary source sample itw-1
			kernel_exec(injectSecondarySource<<<dimGrid, dimBlock>>>(dev_ssLeft[iGpu], dev_ssRight[iGpu], dev_p0[iGpu], it2-1));

			// Damp wavefields
			kernel_exec(dampCosineEdge<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Extract data
			kernel_exec(recordInterpData<<<nblockData, BLOCK_SIZE_DATA>>>(dev_p0[iGpu], dev_dataRegDts[iGpu], its, it2, dev_receiversPositionReg[iGpu]));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		// Switch pointers for secondary source
		dev_ssTemp1[iGpu] = dev_ssLeft[iGpu];
		dev_ssLeft[iGpu] = dev_ssRight[iGpu];
		dev_ssRight[iGpu] = dev_ssTemp1[iGpu];
		cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));
	}

	// Copy data back to host
	cuda_call(hipMemcpy(dataRegDts, dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double), hipMemcpyDeviceToHost));

	/******************************* Deallocation ***************************************/
	// Deallocate all slices
    cuda_call(hipFree(dev_dataRegDts[iGpu]));
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_sourcesSignals[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));

}
void BornShotsFwdGpuWavefield(double *model, double *dataRegDts, double *sourcesSignals, int *sourcesPositionReg, int nSourcesReg, int *receiversPositionReg, int nReceiversReg, double *srcWavefieldDts, double *scatWavefieldDts, int iGpu, int iGpuId){

	// Non-extended Born modeling operator (FORWARD)
	// The source wavelet/signals already contain the second time derivative
	// Set device number
	hipSetDevice(iGpuId);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(int)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(int), hipMemcpyHostToDevice));

	// Sources signals
  	cuda_call(hipMalloc((void**) &dev_sourcesSignals[iGpu], nSourcesReg*host_ntw*sizeof(double))); // Allocate sources signals on device
	cuda_call(hipMemcpy(dev_sourcesSignals[iGpu], sourcesSignals, nSourcesReg*host_ntw*sizeof(double), hipMemcpyHostToDevice)); // Copy sources signals on device

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(int)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(int), hipMemcpyHostToDevice));

	// Allocate and initialize data
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double))); // Allocate data at coarse time-sampling on device
  	cuda_call(hipMemset(dev_dataRegDts[iGpu], 0, nReceiversReg*host_nts*sizeof(double))); // Initialize data on device

	// Initialize source wavefield on device
	cuda_call(hipMemset(dev_BornSrcWavefield[iGpu], 0, host_nz*host_nx*host_nts*sizeof(double))); // Initialize wavefield on device

	// Initialize time-slices for time-stepping
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(double)));

   	// Kernel parameters
	int nblockx = (host_nz-2*FAT) / BLOCK_SIZE;
	int nblocky = (host_nx-2*FAT) / BLOCK_SIZE;
	int nblockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	/************************** Source wavefield computation ****************************/

	for (int its = 0; its < host_nts-1; its++){
		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Compute fine time-step index
			int itw = its * host_sub + it2;

			// Step forward
			kernel_exec(stepFwdGpu<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject source
			kernel_exec(injectSource<<<1, nSourcesReg>>>(dev_sourcesSignals[iGpu], dev_p0[iGpu], itw-1, dev_sourcesPositionReg[iGpu]));

			// Damp wavefields
			kernel_exec(dampCosineEdge<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Extract wavefield
			kernel_exec(interpWavefield<<<dimGrid, dimBlock>>>(dev_BornSrcWavefield[iGpu], dev_p0[iGpu], its, it2));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;
		}
	}

	// Copy wavefield back to host
	cuda_call(hipMemcpy(srcWavefieldDts, dev_BornSrcWavefield[iGpu], host_nz*host_nx*host_nts*sizeof(double), hipMemcpyDeviceToHost));

	/************************** Scattered wavefield computation *************************/
	// Initialize time slices on device
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_ssLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));

	// Allocate and copy model
	cuda_call(hipMemcpy(dev_modelBorn[iGpu], model, host_nz*host_nx*sizeof(double), hipMemcpyHostToDevice)); // Copy model (reflectivity) on device

	// Allocate and initialize scattered wavefield on device
	cuda_call(hipMalloc((void**) &dev_BornSecWavefield, host_nz*host_nx*host_nts*sizeof(double))); // Allocate on device
	cuda_call(hipMemset(dev_BornSecWavefield, 0, host_nz*host_nx*host_nts*sizeof(double))); // Initialize wavefield on device

	// Apply both scalings to reflectivity:
	kernel_exec(scaleReflectivity<<<dimGrid, dimBlock>>>(dev_modelBorn[iGpu], dev_reflectivityScale[iGpu], dev_vel2Dtw2[iGpu]));

	// Compute secondary source for first coarse time index (its = 0)
	kernel_exec(imagingFwdGpu<<<dimGrid, dimBlock>>>(dev_modelBorn[iGpu], dev_ssLeft[iGpu], 0, dev_BornSrcWavefield[iGpu]));

	// Start propagating scattered wavefield
	for (int its = 0; its < host_nts-1; its++){

		// Compute secondary source for first coarse time index (its+1)
		kernel_exec(imagingFwdGpu<<<dimGrid, dimBlock>>>(dev_modelBorn[iGpu], dev_ssRight[iGpu], its+1, dev_BornSrcWavefield[iGpu]));

		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Step forward
			kernel_exec(stepFwdGpu<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject secondary source sample itw-1
			kernel_exec(injectSecondarySource<<<dimGrid, dimBlock>>>(dev_ssLeft[iGpu], dev_ssRight[iGpu], dev_p0[iGpu], it2-1));

			// Damp wavefields
			kernel_exec(dampCosineEdge<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Record wavefield
			kernel_exec(interpWavefield<<<dimGrid, dimBlock>>>(dev_BornSecWavefield, dev_p0[iGpu], its, it2));

			// Extract data
			kernel_exec(recordInterpData<<<nblockData, BLOCK_SIZE_DATA>>>(dev_p0[iGpu], dev_dataRegDts[iGpu], its, it2, dev_receiversPositionReg[iGpu]));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;

		}

		// Switch pointers for secondary source
		dev_ssTemp1[iGpu] = dev_ssLeft[iGpu];
		dev_ssLeft[iGpu] = dev_ssRight[iGpu];
		dev_ssRight[iGpu] = dev_ssTemp1[iGpu];
		cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));

	}

	// Copy data back to host
	cuda_call(hipMemcpy(dataRegDts, dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double), hipMemcpyDeviceToHost));

	// Copy scattered wavefield back to host
	cuda_call(hipMemcpy(scatWavefieldDts, dev_BornSecWavefield, host_nz*host_nx*host_nts*sizeof(double), hipMemcpyDeviceToHost));

	/******************************* Deallocation ***************************************/
	// Deallocate all slices
    cuda_call(hipFree(dev_dataRegDts[iGpu]));
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_sourcesSignals[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));
    cuda_call(hipFree(dev_BornSecWavefield));

}

/****************************************************************************************/
/************************************** Born adjoint ************************************/
/****************************************************************************************/
void BornShotsAdjGpu(double *model, double *dataRegDts, double *sourcesSignals, int *sourcesPositionReg, int nSourcesReg, int *receiversPositionReg, int nReceiversReg, double *srcWavefieldDts, double *recWavefieldDts, int iGpu, int iGpuId){

	// Non-extended Born modeling operator (ADJOINT)
	// We assume the source wavelet/signals already contain the second time derivative
	// Set device number
	hipSetDevice(iGpuId);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(int)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(int), hipMemcpyHostToDevice));

	// Sources signals
  	cuda_call(hipMalloc((void**) &dev_sourcesSignals[iGpu], nSourcesReg*host_ntw*sizeof(double))); // Allocate sources signals on device
	cuda_call(hipMemcpy(dev_sourcesSignals[iGpu], sourcesSignals, nSourcesReg*host_ntw*sizeof(double), hipMemcpyHostToDevice)); // Copy sources signals on device

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(int)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(int), hipMemcpyHostToDevice));

	// Initialize source wavefield on device
	cuda_call(hipMemset(dev_BornSrcWavefield[iGpu], 0, host_nz*host_nx*host_nts*sizeof(double))); // Initialize wavefield on device

	// Initialize time-slices for time-stepping
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(double)));

   	// Kernel parameters
	int nblockx = (host_nz-2*FAT) / BLOCK_SIZE;
	int nblocky = (host_nx-2*FAT) / BLOCK_SIZE;
	int nblockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	/************************** Source wavefield computation ****************************/

	for (int its = 0; its < host_nts-1; its++){
		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Compute fine time-step index
			int itw = its * host_sub + it2;

			// Step forward
			kernel_exec(stepFwdGpu<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject source
			kernel_exec(injectSource<<<1, nSourcesReg>>>(dev_sourcesSignals[iGpu], dev_p0[iGpu], itw-1, dev_sourcesPositionReg[iGpu]));

			// Damp wavefields
			kernel_exec(dampCosineEdge<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Extract wavefield
			kernel_exec(interpWavefield<<<dimGrid, dimBlock>>>(dev_BornSrcWavefield[iGpu], dev_p0[iGpu], its, it2));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;
		}
	}

	/************************** Receiver wavefield computation **************************/
	// Initialize time slices on device
  	cuda_call(hipMemset(dev_ssLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(double)));

	// Model
  	cuda_call(hipMemset(dev_modelBorn[iGpu], 0, host_nz*host_nx*sizeof(double))); // Initialize model on device

	// Data
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double))); // Allocate data on device
	cuda_call(hipMemcpy(dev_dataRegDts[iGpu], dataRegDts, nReceiversReg*host_nts*sizeof(double), hipMemcpyHostToDevice)); // Copy data on device

  	// Main loop
	for (int its = host_nts-2; its > -1; its--){

		for (int it2 = host_sub-1; it2 > -1; it2--){

			// Step adjoint in time
			kernel_exec(stepAdjGpu<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject data
			kernel_exec(interpInjectData<<<nblockData, BLOCK_SIZE_DATA>>>(dev_dataRegDts[iGpu], dev_p0[iGpu], its, it2, dev_receiversPositionReg[iGpu]));

			// Damp wavefield
			kernel_exec(dampCosineEdge<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Interpolate and record time slices of receiver wavefield at coarse sampling (no scaling applied yet)
			extractInterpAdjointWavefield<<<dimGrid, dimBlock>>>(dev_ssLeft[iGpu], dev_ssRight[iGpu], dev_p0[iGpu], it2);

			// Switch pointers for time slices at fine time-sampling
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;
		}

		// Apply imaging condition for its+1
  		kernel_exec(imagingAdjGpu<<<dimGrid, dimBlock>>>(dev_modelBorn[iGpu], dev_ssRight[iGpu], dev_BornSrcWavefield[iGpu], its+1));

		// Switch pointers for receiver wavefield before imaging time derivative
		dev_ssTemp1[iGpu] = dev_ssRight[iGpu];
		dev_ssRight[iGpu] = dev_ssLeft[iGpu];
		dev_ssLeft[iGpu] = dev_ssTemp1[iGpu];
  	cuda_call(hipMemset(dev_ssLeft[iGpu], 0, host_nz*host_nx*sizeof(double))); // Reinitialize slice for coarse time-sampling before time derivative

	} // Finished main loop - we still have to compute imaging condition for its=0

	// Apply imaging condition for its=0
  	kernel_exec(imagingAdjGpu<<<dimGrid, dimBlock>>>(dev_modelBorn[iGpu], dev_ssRight[iGpu], dev_BornSrcWavefield[iGpu], 0));

  	// Scale model for finite-difference and secondary source coefficient
	kernel_exec(scaleReflectivity<<<dimGrid, dimBlock>>>(dev_modelBorn[iGpu], dev_reflectivityScale[iGpu], dev_vel2Dtw2[iGpu]));

	// Copy model back to host
	cuda_call(hipMemcpy(model, dev_modelBorn[iGpu], host_nz*host_nx*sizeof(double), hipMemcpyDeviceToHost));

	/******************************* Deallocation ***************************************/
	// Deallocate all slices
    cuda_call(hipFree(dev_dataRegDts[iGpu]));
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_sourcesSignals[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));

}
void BornShotsAdjGpuWavefield(double *model, double *dataRegDts, double *sourcesSignals, int *sourcesPositionReg, int nSourcesReg, int *receiversPositionReg, int nReceiversReg, double *srcWavefieldDts, double *recWavefieldDts, int iGpu, int iGpuId){

	// Non-extended Born modeling operator (ADJOINT)
	// We assume the source wavelet/signals already contain the second time derivative
	// Set device number
	hipSetDevice(iGpuId);

	// Sources geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nSourcesReg), &nSourcesReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_sourcesPositionReg[iGpu], nSourcesReg*sizeof(int)));
	cuda_call(hipMemcpy(dev_sourcesPositionReg[iGpu], sourcesPositionReg, nSourcesReg*sizeof(int), hipMemcpyHostToDevice));

	// Sources signals
  	cuda_call(hipMalloc((void**) &dev_sourcesSignals[iGpu], nSourcesReg*host_ntw*sizeof(double))); // Allocate sources signals on device
	cuda_call(hipMemcpy(dev_sourcesSignals[iGpu], sourcesSignals, nSourcesReg*host_ntw*sizeof(double), hipMemcpyHostToDevice)); // Copy sources signals on device

	// Receivers geometry
	cuda_call(hipMemcpyToSymbol(HIP_SYMBOL(dev_nReceiversReg), &nReceiversReg, sizeof(int), 0, hipMemcpyHostToDevice));
	cuda_call(hipMalloc((void**) &dev_receiversPositionReg[iGpu], nReceiversReg*sizeof(int)));
	cuda_call(hipMemcpy(dev_receiversPositionReg[iGpu], receiversPositionReg, nReceiversReg*sizeof(int), hipMemcpyHostToDevice));

	// Initialize source wavefield on device
	cuda_call(hipMemset(dev_BornSrcWavefield[iGpu], 0, host_nz*host_nx*host_nts*sizeof(double))); // Initialize wavefield on device

	// Initialize time-slices for time-stepping
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(double)));

   	// Kernel parameters
	int nblockx = (host_nz-2*FAT) / BLOCK_SIZE;
	int nblocky = (host_nx-2*FAT) / BLOCK_SIZE;
	int nblockData = (nReceiversReg+BLOCK_SIZE_DATA-1) / BLOCK_SIZE_DATA;
	dim3 dimGrid(nblockx, nblocky);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	/************************** Source wavefield computation ****************************/

	for (int its = 0; its < host_nts-1; its++){
		for (int it2 = 1; it2 < host_sub+1; it2++){

			// Compute fine time-step index
			int itw = its * host_sub + it2;

			// Step forward
			kernel_exec(stepFwdGpu<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject source
			kernel_exec(injectSource<<<1, nSourcesReg>>>(dev_sourcesSignals[iGpu], dev_p0[iGpu], itw-1, dev_sourcesPositionReg[iGpu]));

			// Damp wavefields
			kernel_exec(dampCosineEdge<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Extract wavefield
			kernel_exec(interpWavefield<<<dimGrid, dimBlock>>>(dev_BornSrcWavefield[iGpu], dev_p0[iGpu], its, it2));

			// Switch pointers
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;
		}
	}

	/************************** Receiver wavefield computation **************************/
	// Initialize time slices on device
  	cuda_call(hipMemset(dev_ssLeft[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_ssRight[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_p0[iGpu], 0, host_nz*host_nx*sizeof(double)));
  	cuda_call(hipMemset(dev_p1[iGpu], 0, host_nz*host_nx*sizeof(double)));

	// Allocate and initialize receiver wavefield on device
	cuda_call(hipMalloc((void**) &dev_BornSecWavefield, host_nz*host_nx*host_nts*sizeof(double))); // Allocate on device
	cuda_call(hipMemset(dev_BornSecWavefield, 0, host_nz*host_nx*host_nts*sizeof(double))); // Initialize wavefield on device

	// Model
  	cuda_call(hipMemset(dev_modelBorn[iGpu], 0, host_nz*host_nx*sizeof(double))); // Initialize model on device

	// Data
  	cuda_call(hipMalloc((void**) &dev_dataRegDts[iGpu], nReceiversReg*host_nts*sizeof(double))); // Allocate data on device
	cuda_call(hipMemcpy(dev_dataRegDts[iGpu], dataRegDts, nReceiversReg*host_nts*sizeof(double), hipMemcpyHostToDevice)); // Copy data on device

  	// Main loop
	for (int its = host_nts-2; its > -1; its--){

		for (int it2 = host_sub-1; it2 > -1; it2--){

			// Step adjoint in time
			kernel_exec(stepAdjGpu<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu], dev_p0[iGpu], dev_vel2Dtw2[iGpu]));

			// Inject data
			kernel_exec(interpInjectData<<<nblockData, BLOCK_SIZE_DATA>>>(dev_dataRegDts[iGpu], dev_p0[iGpu], its, it2, dev_receiversPositionReg[iGpu]));

			// Damp wavefield
			kernel_exec(dampCosineEdge<<<dimGrid, dimBlock>>>(dev_p0[iGpu], dev_p1[iGpu]));

			// Interpolate and record time slices of receiver wavefield at coarse sampling (no scaling applied yet)
			extractInterpAdjointWavefield<<<dimGrid, dimBlock>>>(dev_ssLeft[iGpu], dev_ssRight[iGpu], dev_p0[iGpu], it2);

			// Switch pointers for time slices at fine time-sampling
			dev_temp1[iGpu] = dev_p0[iGpu];
			dev_p0[iGpu] = dev_p1[iGpu];
			dev_p1[iGpu] = dev_temp1[iGpu];
			dev_temp1[iGpu] = NULL;
		}

		// Apply imaging condition for its+1
  		kernel_exec(imagingAdjGpu<<<dimGrid, dimBlock>>>(dev_modelBorn[iGpu], dev_ssRight[iGpu], dev_BornSrcWavefield[iGpu], its+1));

		// Record and scale receiver wavefield at coarse sampling for its+1
		kernel_exec(recordScaleWavefield<<<dimGrid, dimBlock>>>(dev_BornSecWavefield, dev_ssRight[iGpu], its+1, dev_reflectivityScale[iGpu], dev_vel2Dtw2[iGpu]));

		// Switch pointers for receiver wavefield before imaging time derivative
		dev_ssTemp1[iGpu] = dev_ssRight[iGpu];
		dev_ssRight[iGpu] = dev_ssLeft[iGpu];
		dev_ssLeft[iGpu] = dev_ssTemp1[iGpu];
  		cuda_call(hipMemset(dev_ssLeft[iGpu], 0, host_nz*host_nx*sizeof(double))); // Reinitialize slice for coarse time-sampling before time derivative

	} // Finished main loop - we still have to compute imaging condition for its=0

	// Save receiver wavefield at its=0
	kernel_exec(recordScaleWavefield<<<dimGrid, dimBlock>>>(dev_BornSecWavefield, dev_ssRight[iGpu], 0, dev_reflectivityScale[iGpu], dev_vel2Dtw2[iGpu]));

	// Apply imaging condition for its=0
  	kernel_exec(imagingAdjGpu<<<dimGrid, dimBlock>>>(dev_modelBorn[iGpu], dev_ssRight[iGpu], dev_BornSrcWavefield[iGpu], 0));

  	// Scale model for finite-difference and secondary source coefficient
	// It's better to apply it once and for all than at every time-steps
	kernel_exec(scaleReflectivity<<<dimGrid, dimBlock>>>(dev_modelBorn[iGpu], dev_reflectivityScale[iGpu], dev_vel2Dtw2[iGpu]));

	// Copy model back to host
	cuda_call(hipMemcpy(model, dev_modelBorn[iGpu], host_nz*host_nx*sizeof(double), hipMemcpyDeviceToHost));

	// Copy scattered wavefield back to host
	cuda_call(hipMemcpy(recWavefieldDts, dev_BornSecWavefield, host_nz*host_nx*host_nts*sizeof(double), hipMemcpyDeviceToHost));

	/******************************* Deallocation ***************************************/
	// Deallocate all slices
    cuda_call(hipFree(dev_dataRegDts[iGpu]));
    cuda_call(hipFree(dev_sourcesPositionReg[iGpu]));
    cuda_call(hipFree(dev_sourcesSignals[iGpu]));
    cuda_call(hipFree(dev_receiversPositionReg[iGpu]));
    cuda_call(hipFree(dev_BornSecWavefield));
}
